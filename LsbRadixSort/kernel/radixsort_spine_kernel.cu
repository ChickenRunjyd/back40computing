#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * 
 * 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 * 
 ******************************************************************************/


/******************************************************************************
 * Top-level histogram/spine scanning kernel
 ******************************************************************************/

#pragma once

#include "radixsort_kernel_common.cu"

namespace b40c {



/******************************************************************************
 * Scans a cycle of RADIXSORT_CYCLE_ELEMENTS elements 
 ******************************************************************************/

template<
	unsigned int SMEM_ROWS,
	unsigned int RAKING_THREADS,
	unsigned int PARTIALS_PER_ROW,
	unsigned int PARTIALS_PER_SEG>
__device__ __forceinline__ void SrtsScanCycle(
	unsigned int smem[SMEM_ROWS][PARTIALS_PER_ROW + 1],
	unsigned int *smem_offset,
	unsigned int *smem_segment,
	unsigned int warpscan[2][WARP_THREADS],
	uint4 *in, 
	uint4 *out,
	unsigned int &carry)
{
	uint4 datum; 

	// read input data
	datum = in[threadIdx.x];

	smem_offset[0] = datum.x + datum.y + datum.z + datum.w;

	__syncthreads();

	if (threadIdx.x < WARP_THREADS) {

		unsigned int partial_reduction = SerialReduce<PARTIALS_PER_SEG>(smem_segment);

		unsigned int seed = WarpScan<WARP_THREADS, false>(warpscan, partial_reduction, 0);
		seed += carry;		
		carry += warpscan[1][WARP_THREADS - 1];	
		
		SerialScan<PARTIALS_PER_SEG>(smem_segment, seed);
	}

	__syncthreads();

	unsigned int part0 = smem_offset[0];
	unsigned int part1;

	part1 = datum.x + part0;
	datum.x = part0;
	part0 = part1 + datum.y;
	datum.y = part1;

	part1 = datum.z + part0;
	datum.z = part0;
	part0 = part1 + datum.w;
	datum.w = part1;
	
	out[threadIdx.x] = datum;
}


/******************************************************************************
 * Spine/histogram Scan Kernel Entry Point
 ******************************************************************************/

template <typename T>
__global__ void SrtsScanSpine(
	unsigned int *d_ispine,
	unsigned int *d_ospine,
	unsigned int normal_block_elements)
{
	const unsigned int LOG_RAKING_THREADS 		= LOG_WARP_THREADS;				
	const unsigned int RAKING_THREADS 			= 1 << LOG_RAKING_THREADS;		
	
	const unsigned int LOG_PARTIALS				= RADIXSORT_LOG_THREADS;				
	const unsigned int PARTIALS			 		= 1 << LOG_PARTIALS;
	
	const unsigned int LOG_PARTIALS_PER_SEG 	= LOG_PARTIALS - LOG_RAKING_THREADS;	
	const unsigned int PARTIALS_PER_SEG 		= 1 << LOG_PARTIALS_PER_SEG;

	const unsigned int LOG_PARTIALS_PER_ROW		= (LOG_PARTIALS_PER_SEG < LOG_MEM_BANKS(__CUDA_ARCH__)) ? LOG_MEM_BANKS(__CUDA_ARCH__) : LOG_PARTIALS_PER_SEG;		// floor of 32 elts per row
	const unsigned int PARTIALS_PER_ROW			= 1 << LOG_PARTIALS_PER_ROW;
	
	const unsigned int LOG_SEGS_PER_ROW 		= LOG_PARTIALS_PER_ROW - LOG_PARTIALS_PER_SEG;	
	const unsigned int SEGS_PER_ROW				= 1 << LOG_SEGS_PER_ROW;

	const unsigned int SMEM_ROWS 				= PARTIALS / PARTIALS_PER_ROW;
	
	__shared__ unsigned int smem[SMEM_ROWS][PARTIALS_PER_ROW + 1];
	__shared__ unsigned int warpscan[2][WARP_THREADS];

	unsigned int *smem_segment;
	unsigned int carry;

	unsigned int row = threadIdx.x >> LOG_PARTIALS_PER_ROW;		
	unsigned int col = threadIdx.x & (PARTIALS_PER_ROW - 1);			
	unsigned int *smem_offset = &smem[row][col];

	if (blockIdx.x > 0) {
		return;
	}
	
	if (threadIdx.x < RAKING_THREADS) {
		
		// two segs per row, odd segs are offset by 8
		row = threadIdx.x >> LOG_SEGS_PER_ROW;
		col = (threadIdx.x & (SEGS_PER_ROW - 1)) << LOG_PARTIALS_PER_SEG;
		smem_segment = &smem[row][col];
	
		if (threadIdx.x < WARP_THREADS) {
			carry = 0;
			warpscan[0][threadIdx.x] = 0;
		}
	}

	// scan the spine in blocks of cycle_elements
	unsigned int block_offset = 0;
	while (block_offset < normal_block_elements) {
		
		SrtsScanCycle<SMEM_ROWS, RAKING_THREADS, PARTIALS_PER_ROW, PARTIALS_PER_SEG>(	
			smem, smem_offset, smem_segment, warpscan,
			(uint4 *) &d_ispine[block_offset], 
			(uint4 *) &d_ospine[block_offset], 
			carry);

		block_offset += RADIXSORT_SPINE_CYCLE_ELEMENTS;
	}
} 


} // namespace b40c

