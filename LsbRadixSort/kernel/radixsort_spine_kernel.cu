#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 * 
 ******************************************************************************/


/******************************************************************************
 * Top-level histogram/spine scanning kernel. The second kernel in a 
 * radix-sorting digit-place pass. 
 ******************************************************************************/

#pragma once

#include "radixsort_kernel_common.cu"

namespace b40c {
namespace lsb_radix_sort {
namespace scan {


/******************************************************************************
 * Granularity Configuration
 ******************************************************************************/

/**
 * Spine-scan granularity configuration.  This C++ type encapsulates our 
 * kernel-tuning parameters (they are reflected via the static fields).
 *  
 * The kernels are specialized for problem-type, SM-version, etc. by declaring 
 * them with different performance-tuned parameterizations of this type.  By 
 * incorporating this type into the kernel code itself, we guide the compiler in 
 * expanding/unrolling the kernel code for specific architectures and problem 
 * types.    
 */
template <
	typename _ScanType,
	typename _IndexType,
	int _CTA_OCCUPANCY,
	int _LOG_THREADS,
	int _LOG_LOAD_VEC_SIZE,
	int _LOG_LOADS_PER_TILE,
	int _LOG_RAKING_THREADS,
	CacheModifier _CACHE_MODIFIER>

struct ScanConfig
{
	typedef _ScanType							ScanType;
	typedef _IndexType							IndexType;
	static const int CTA_OCCUPANCY  			= _CTA_OCCUPANCY;
	static const int LOG_THREADS 				= _LOG_THREADS;
	static const int LOG_LOAD_VEC_SIZE  		= _LOG_LOAD_VEC_SIZE;
	static const int LOG_LOADS_PER_TILE 		= _LOG_LOADS_PER_TILE;
	static const int LOG_RAKING_THREADS			= _LOG_RAKING_THREADS;
	static const CacheModifier CACHE_MODIFIER 	= _CACHE_MODIFIER;
};



/******************************************************************************
 * Kernel Configuration  
 ******************************************************************************/

/**
 * A detailed upsweep configuration type that specializes kernel code for a specific 
 * sorting pass.  It encapsulates granularity details derived from the inherited 
 * UpsweepConfigType 
 */
template <typename ScanConfigType>
struct ScanKernelConfig : ScanConfigType
{
	static const int THREADS						= 1 << ScanConfigType::LOG_THREADS;
	
	static const int LOG_WARPS						= ScanConfigType::LOG_THREADS - B40C_LOG_WARP_THREADS(__B40C_CUDA_ARCH__);
	static const int WARPS							= 1 << LOG_WARPS;	
	
	static const int LOAD_VEC_SIZE					= 1 << ScanConfigType::LOG_LOAD_VEC_SIZE;
	static const int LOADS_PER_TILE					= 1 << ScanConfigType::LOG_LOADS_PER_TILE;

	static const int LOG_TILE_ELEMENTS				= ScanConfigType::LOG_THREADS + 
															ScanConfigType::LOG_LOADS_PER_TILE +
															ScanConfigType::LOG_LOAD_VEC_SIZE;
	static const int TILE_ELEMENTS					= 1 << LOG_TILE_ELEMENTS;
	
	// We reduce/scan the elements of a loaded vector in registers, and then place that  
	// partial reduction into smem rows for further reduction/scanning
	
	// We need a two-level grid if (LOG_RAKING_THREADS > LOG_WARP_THREADS).  If so, we 
	// back up the primary raking warps with a single warp of raking-threads.
	static const bool TwoLevelGrid 					= (ScanConfigType::LOG_RAKING_THREADS > B40C_LOG_WARP_THREADS(__B40C_CUDA_ARCH__));

	// Primary smem SRTS grid type
	typedef SrtsGrid<
		typename ScanConfigType::ScanType,
		ScanConfigType::LOG_THREADS,
		ScanConfigType::LOG_LOADS_PER_TILE, 
		ScanConfigType::LOG_RAKING_THREADS> PrimaryGrid;
	
	// Secondary smem SRTS grid type
	typedef SrtsGrid<
		typename ScanConfigType::ScanType,
		ScanConfigType::LOG_RAKING_THREADS,
		0, 
		B40C_LOG_WARP_THREADS(__B40C_CUDA_ARCH__)> SecondaryGrid;
	
		
	static const int SMEM_BYTES						= (TwoLevelGrid) ? 
															PrimaryGrid::SMEM_BYTES + SecondaryGrid::SMEM_BYTES :	// two-level smem SRTS 
															PrimaryGrid::SMEM_BYTES;								// one-level smem SRTS
};
	
	
	


/******************************************************************************
 * Spine-scan kernel subroutines
 ******************************************************************************/


// Reduce each load in registers and place into smem
template <typename Config> 
struct ReduceVectors
{
	typedef typename Config::ScanType ScanType;
	
	// Iterate over vec-elements
	template <int LOAD, int VEC, int __dummy = 0>
	struct Iterate {
		static __device__ __forceinline__ void Invoke(
			ScanType partial,
			ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
			ScanType *base_partial) 
		{
			partial += data[LOAD][VEC];
			Iterate<LOAD, VEC + 1>::Invoke(partial, data, base_partial);
		}
	};

	// First vector element: Identity
	template <int LOAD, int __dummy>
	struct Iterate<LOAD, 0, __dummy> {
		static __device__ __forceinline__ void Invoke(
			ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
			ScanType *base_partial) 
		{
			Iterate<LOAD, 1>::Invoke(data[LOAD][0], data, base_partial);
		}
	};

	// Last vector element + 1: Next load
	template <int LOAD, int __dummy>
	struct Iterate<LOAD, Config::LOAD_VEC_SIZE, __dummy> {
		static __device__ __forceinline__ void Invoke(
			ScanType partial,
			ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
			ScanType *base_partial) 
		{
			// Store partial reduction into SRTS grid
			base_partial[LOAD * Config::PrimaryGrid::PARTIAL_STRIDE] = partial;

			// Next load
			Iterate<LOAD + 1, 0>::Invoke(data, base_partial);
		}
	};
	
	// Last load + 1: Terminate
	template <int __dummy>
	struct Iterate<Config::LOADS_PER_TILE, 0, __dummy> {
		static __device__ __forceinline__ void Invoke(
			ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
			ScanType *base_partial) {} 
	};
	
	// Interface
	static __device__ __forceinline__ void Invoke(
		ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
		ScanType *base_partial)
	{
		Iterate<0, 0>::template Invoke(data, base_partial);
	}

};


// Scan each load in registers, seeding from smem partials
template <typename Config> 
struct ScanVectors
{
	typedef typename Config::ScanType ScanType;
	
	// Iterate over vec-elements
	template <int LOAD, int VEC, int __dummy = 0>
	struct Iterate {
		static __device__ __forceinline__ void Invoke(
			ScanType exclusive_partial,
			ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
			ScanType *base_partial) 
		{
			ScanType inclusive_partial = data[LOAD][VEC] + exclusive_partial;
			data[LOAD][VEC] = exclusive_partial;
			Iterate<LOAD, VEC + 1>::Invoke(inclusive_partial, data, base_partial);
		}
	};

	// First vector element: Load exclusive partial reduction from SRTS grid
	template <int LOAD, int __dummy>
	struct Iterate<LOAD, 0, __dummy> {
		static __device__ __forceinline__ void Invoke(
			ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
			ScanType *base_partial) 
		{
			ScanType exclusive_partial = base_partial[LOAD * Config::PrimaryGrid::PARTIAL_STRIDE];
			ScanType inclusive_partial = data[LOAD][0] + exclusive_partial;
			data[LOAD][0] = exclusive_partial;
			Iterate<LOAD, 1>::Invoke(inclusive_partial, data, base_partial);
		}
	};

	// Last vector element + 1: Next load
	template <int LOAD, int __dummy>
	struct Iterate<LOAD, Config::LOAD_VEC_SIZE, __dummy> {
		static __device__ __forceinline__ void Invoke(
			ScanType exclusive_partial,
			ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
			ScanType *base_partial) 
		{
			// Next load
			Iterate<LOAD + 1, 0>::Invoke(data, base_partial);
		}
	};
	
	// Last load + 1: Terminate
	template <int __dummy>
	struct Iterate<Config::LOADS_PER_TILE, 0, __dummy> {
		static __device__ __forceinline__ void Invoke(
			ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
			ScanType *base_partial) {} 
	};
	
	// Interface
	static __device__ __forceinline__ void Invoke(
		ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE], 
		ScanType *base_partial)
	{
		Iterate<0, 0>::template Invoke(data, base_partial);
	}

};



/**
 * Warp rake and scan. Must hold that the number of raking threads in the grid 
 * config type is at most the size of a warp.  (May be less.)
 */
template <typename Grid> 
__device__ __forceinline__ void WarpRakeAndScan(
	typename Grid::PartialType 	*raking_seg,
	typename Grid::PartialType 	warpscan[2][Grid::RAKING_THREADS],
	typename Grid::PartialType 	&carry)
{
	typedef typename Grid::PartialType PartialType;
	
	if (threadIdx.x < Grid::RAKING_THREADS) {
		
		// Raking reduction  
		PartialType partial = SerialReduce<PartialType, Grid::PARTIALS_PER_SEG>::Invoke(raking_seg);
		
		// Warpscan
		PartialType warpscan_total;
		partial = WarpScan<PartialType, Grid::RAKING_THREADS>::Invoke(partial, warpscan_total, warpscan);
		partial += carry;
		carry += warpscan_total;			// Increment the CTA's running total by the full tile reduction

		// Raking scan 
		SerialScan<PartialType, Grid::PARTIALS_PER_SEG>::Invoke(raking_seg, partial);
	}
}


/**
 * Process a scan tile.
 */
template <typename Config, bool TwoLevelGrid> struct ProcessTile;


/**
 * Process a scan tile using only a one-level raking grid.  (One warp or smaller of raking threads.)
 */
template <typename Config> 
struct ProcessTile <Config, false>
{
	typedef typename Config::ScanType ScanType;
	typedef typename Config::IndexType IndexType;
	
	__device__ __forceinline__ static void Invoke(
		ScanType 	*primary_base_partial,
		ScanType 	*primary_raking_seg,
		ScanType 	*secondary_base_partial,
		ScanType 	*secondary_raking_seg,
		ScanType 	warpscan[2][Config::PrimaryGrid::RAKING_THREADS],
		ScanType 	*d_data,
		IndexType 	cta_offset,
		ScanType 	&carry)
	{
		// Tile of scan elements
		ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE];
		
		// Load tile
		LoadTile<ScanType, IndexType, Config::LOG_LOADS_PER_TILE, Config::LOG_LOAD_VEC_SIZE, Config::THREADS, Config::CACHE_MODIFIER, true>::Invoke(
			data, d_data, cta_offset);
		
		// Reduce in registers, place partials in smem
		ReduceVectors<Config>::Invoke(data, primary_base_partial);
		
		__syncthreads();
		
		// Primary rake and scan (guaranteed one warp or fewer raking threads)
		WarpRakeAndScan<Config::PrimaryGrid>(primary_raking_seg, warpscan, carry);
		
		__syncthreads();

		// Extract partials from smem, scan in registers
		ScanVectors<Config>::Invoke(data, primary_base_partial);
		
		// Store tile
		StoreTile<ScanType, IndexType, Config::LOG_LOADS_PER_TILE, Config::LOG_LOAD_VEC_SIZE, Config::THREADS, Config::CACHE_MODIFIER, true>::Invoke(
			data, d_data, cta_offset);
	}
};


/**
 * Process a scan tile using a two-level raking grid.  (More than one warp of raking threads.)
 */
template <typename Config> 
struct ProcessTile <Config, true>
{
	typedef typename Config::ScanType ScanType;
	typedef typename Config::IndexType IndexType;
	
	__device__ __forceinline__ static void Invoke(
		ScanType 	*primary_base_partial,
		ScanType 	*primary_raking_seg,
		ScanType 	*secondary_base_partial,
		ScanType 	*secondary_raking_seg,
		ScanType 	warpscan[2][Config::SecondaryGrid::RAKING_THREADS],
		ScanType 	*d_data,
		IndexType 	cta_offset,
		ScanType 	&carry)
	{
		// Tile of scan elements
		ScanType data[Config::LOADS_PER_TILE][Config::LOAD_VEC_SIZE];
		
		// Load tile
		LoadTile<ScanType, IndexType, Config::LOG_LOADS_PER_TILE, Config::LOG_LOAD_VEC_SIZE, Config::THREADS, Config::CACHE_MODIFIER, true>::Invoke(
			data, d_data, cta_offset);
		
		// Reduce in registers, place partials in smem
		ReduceVectors<Config>::Invoke(data, primary_base_partial);
		
		__syncthreads();
		
		// Raking reduction in primary grid, place result partial into secondary grid
		if (threadIdx.x < Config::PrimaryGrid::RAKING_THREADS) {
			ScanType partial = SerialReduce<ScanType, Config::PrimaryGrid::PARTIALS_PER_SEG>::Invoke(primary_raking_seg);
			*secondary_base_partial = partial;
		}

		__syncthreads();
		
		// Secondary rake and scan (guaranteed one warp or fewer raking threads)
		WarpRakeAndScan<Config::SecondaryGrid>(secondary_raking_seg, warpscan, carry);
		
		__syncthreads();

		// Raking scan in primary grid seeded by partial from secondary grid
		if (threadIdx.x < Config::PrimaryGrid::RAKING_THREADS) {
			ScanType partial = *secondary_base_partial;
			SerialScan<ScanType, Config::PrimaryGrid::PARTIALS_PER_SEG>::Invoke(primary_raking_seg, partial);
		}

		__syncthreads();

		// Extract partials from smem, scan in registers
		ScanVectors<Config>::Invoke(data, primary_base_partial);
		
		// Store tile
		StoreTile<ScanType, IndexType, Config::LOG_LOADS_PER_TILE, Config::LOG_LOAD_VEC_SIZE, Config::THREADS, Config::CACHE_MODIFIER, true>::Invoke(
			data, d_data, cta_offset);
		
	}
};


/**
 * Host stub to calm the linker for arch-specializations that we didn't 
 * end up compiling PTX for.
 */
template <typename KernelConfig> 
__host__ void __wrapper__device_stub_LsbSpineScanKernel(
	typename KernelConfig::ScanType *&, 
	typename KernelConfig::IndexType &) {}


/**
 * Kernel entry point
 */
template <typename KernelConfig>
__launch_bounds__ (KernelConfig::THREADS, KernelConfig::CTA_OCCUPANCY)
__global__ void LsbSpineScanKernel(
	typename KernelConfig::ScanType *d_spine,
	typename KernelConfig::IndexType spine_elements)
{
	typedef typename KernelConfig::ScanType ScanType;
	typedef typename KernelConfig::IndexType IndexType;

	// Shared memory pool
	__shared__ unsigned char smem_pool[KernelConfig::SMEM_BYTES];
	__shared__ int warpscan[2][B40C_WARP_THREADS(__B40C_CUDA_ARCH__)];
	
	// Exit if we're not the first CTA
	if (blockIdx.x > 0) {
		return;
	}
	
	ScanType 	*primary_grid = reinterpret_cast<ScanType*>(smem_pool);
	ScanType 	*primary_base_partial = KernelConfig::PrimaryGrid::BasePartial(primary_grid);
	ScanType 	*primary_raking_seg = 0;

	ScanType 	*secondary_base_partial = 0;
	ScanType 	*secondary_raking_seg = 0;
	
	ScanType carry = 0;
	
	// Initialize partial-placement and raking offset pointers
	if (threadIdx.x < KernelConfig::PrimaryGrid::RAKING_THREADS) {

		primary_raking_seg = KernelConfig::PrimaryGrid::RakingSegment(primary_grid);

		ScanType *secondary_grid = reinterpret_cast<ScanType*>(smem_pool + KernelConfig::PrimaryGrid::SMEM_BYTES);		// Offset by the primary grid
		secondary_base_partial = KernelConfig::SecondaryGrid::BasePartial(secondary_grid);
		if (KernelConfig::TwoLevelGrid && (threadIdx.x < KernelConfig::SecondaryGrid::RAKING_THREADS)) {
			secondary_raking_seg = KernelConfig::SecondaryGrid::RakingSegment(secondary_grid);
		}

		// Initialize warpscan
		if (threadIdx.x < B40C_WARP_THREADS(__B40C_CUDA_ARCH__)) {
			warpscan[0][threadIdx.x] = 0;
		}
	}

	// Scan the spine in tiles
	IndexType cta_offset = 0;
	while (cta_offset < spine_elements) {
		
		ProcessTile<KernelConfig, KernelConfig::TwoLevelGrid>::Invoke(	
			primary_base_partial,
			primary_raking_seg,
			secondary_base_partial,
			secondary_raking_seg,
			warpscan,
			d_spine,
			cta_offset,
			carry);

		cta_offset += KernelConfig::TILE_ELEMENTS;
	}
} 


} // namespace scan
} // namespace lsb_radix_sort
} // namespace b40c

