#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * 
 * 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 * 
 ******************************************************************************/


/******************************************************************************
 * 
 * Advanced test driver program for SRTS Radix Sorting
 *
 * WARNING: This program assumes knowlege of the temporary storage management
 * needed for performing SRTS radix sort -- do not use it as a reference for 
 * embedding SRTS sorting within your application.  See the Simple test driver 
 * program instead.
 * 
 ******************************************************************************/

#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <math.h> 
#include <float.h>

#include <radixsort_api.cu>			// Sorting includes
#include "test_utils.cu"			// Utilities and correctness-checking
#include "inc/cutil.h"				// Utilities for commandline parsing

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_verbose2;
bool g_verify;
int  g_entropy_reduction = 0;
bool g_regen;


/******************************************************************************
 * Empty Kernels
 ******************************************************************************/

/**
 * Dummy kernel to demarcate iterations of the same problem size in the profiler logs 
 */
__global__ void DummyKernel()
{
}



/******************************************************************************
 * Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\nsrts_radix_sort [--device=<device index>] [--v[2]] [--noverify]\n");
	printf("[--i=<num-iterations>] [--entropy-reduction=<level>] [--regen]\n");
	printf("[--key-bytes=<1|2|4|8>] [--value-bytes=<0|4|8|16>]\n");
	printf("[--n=<num-elements> | --n-input=<num-elements listfile>]\n");
	printf("[--max-blocks=<max-thread-blocks> | --max-blocks-input=<max-thread-blocks listfile>]\n");
	printf("\n");
	printf("\t--v\tDisplays kernel launch config info.\n");
	printf("\n");
	printf("\t--v2\tSame as --v, but displays the sorted keys to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the sorting operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
	printf("\t--n-input\tA file of problem sizes, one per line.\n");
	printf("\n");
	printf("\t--max-blocks\tThe maximum number of threadblocks to launch.\n");
	printf("\t\t\tDefault = -1 (i.e., the API will select an appropriate value)\n");
	printf("\n");
	printf("\t--max-blocks-input\tA file of maximum threadblocks, one per line.\n");
	printf("\n");
	printf("\t--key-bytes\tThe number of key bytes to use.  Default=4\n");
	printf("\n");
	printf("\t--value-bytes\tThe number of value satellite bytes to pair with\n");
	printf("\t\t\tthe key.  Default=0 (I.e., keys-only)\n");
	printf("\n");
	printf("[\t--entropy-reduction=<level>\tSpecifies the number of bitwise-AND'ing\n");
	printf("\t\t\titerations for random key data.  Default = 0, Identical keys = -1\n");
	printf("\n");
	printf("[\t--regen\tGenerates new random numbers for every problem size \n");
	printf("\t\t\tin <num-elements-listfile>\n");
	printf("\n");
	printf("\t--noverify\tSpecifies that results should not be copied back and checked for correctness\n");
	printf("\n");
}


/**
 * Reads a newline-separated list of numbers from an input file.
 * Allocates memory for the returned list.
 */
void ReadList(
	int* &list, 
	unsigned int &len, 
	char* filename, 
	unsigned int default_val) 
{
	if (filename == NULL) {
		len = 1;
		list = (int*) malloc(len * sizeof(int));
		list[0] = default_val;
		return;
	}

	unsigned int data;
	FILE* fin = fopen(filename, "r");
	if (fin == NULL) {
		fprintf(stderr, "Could not open file.  Exiting.\n");
		exit(1);
	}
	len = 0;

	while(fscanf(fin, "%d\n", &data) > 0) {
		len++;
	}

	list = (int*) malloc(len * sizeof(int));
	rewind(fin);
	len = 0;

	while(fscanf(fin, "%d\n", &data) > 0) {
	
		list[len] = data;
		len++;
	}

	fclose(fin);
}




/**
 * Uses the GPU to sort the specified vector of elements for the given 
 * number of iterations, displaying runtime information.
 */
template <typename K, typename V, bool KEYS_ONLY>
void TimedSort(
	unsigned int num_elements, 
	unsigned int max_grid_size,
	K *h_keys,
	K *h_keys_result,
	RadixSortStorage<K, V>	&device_storage,
	unsigned int iterations) 
{
	// Create and initialize sorting enactor
	RadixSortingEnactor<K, V> sorting_enactor(num_elements, max_grid_size);

	CUT_CHECK_ERROR("Kernel execution failed (errors before launch)");

	// Create timing records
	hipEvent_t start_event, stop_event;
	CUDA_SAFE_CALL( hipEventCreate(&start_event) );
	CUDA_SAFE_CALL( hipEventCreate(&stop_event) );

	// Perform the timed number of sorting iterations
	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < iterations; i++) {

		RADIXSORT_DEBUG = (g_verbose && (i == 0));

		// Move a fresh copy of the problem into device storage
		CUDA_SAFE_CALL( hipMemcpy(device_storage.d_keys, h_keys, num_elements * sizeof(K), hipMemcpyHostToDevice) );

		// Start cuda timing record
		CUDA_SAFE_CALL( hipEventRecord(start_event, 0) );

		// Call the sorting API routine
		hipError_t retval = sorting_enactor.EnactSort(device_storage);

		// End cuda timing record
		CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
		CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
		CUDA_SAFE_CALL( hipEventElapsedTime(&duration, start_event, stop_event));
		elapsed += (double) duration;
		
		if (i == 0) {
			printf("%d-byte keys, %d-byte values, %d iterations, %d elements, %d max grid size", 
				sizeof(K), 
				(KEYS_ONLY) ? 0 : sizeof(V),
				iterations, 
				num_elements,
				max_grid_size);
			fflush(stdout);
		}
	}

	// Display timing information
	double avg_runtime = elapsed / iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0; 
    printf(", %f GPU ms, %f x10^9 elts/sec\n", 
		avg_runtime,
		throughput);

    // Clean up events
	CUDA_SAFE_CALL( hipEventDestroy(start_event) );
	CUDA_SAFE_CALL( hipEventDestroy(stop_event) );
	
	// Copy out sorted keys and check
    if (g_verify || g_verbose) {

    	CUDA_SAFE_CALL( hipMemcpy(h_keys_result, device_storage.d_keys, num_elements * sizeof(K), hipMemcpyDeviceToHost) );

		// Display sorted key data
		if (g_verbose2) {
			printf("\n\nKeys:\n");
			for (int i = 0; i < num_elements; i++) {	
				PrintValue<K>(h_keys_result[i]);
				printf(", ");
			}
			printf("\n\n");
		}	
		
	    // Verify solution
		if (g_verify) {
			VerifySort<K>(h_keys_result, num_elements, true);
			printf("\n");
			fflush(stdout);
		}
    }
	
}


/**
 * Creates an example sorting problem whose keys is a vector of the specified 
 * number of K elements, values of V elements, and then dispatches the problem 
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<typename K, typename V, bool KEYS_ONLY>
void TestSort(
	unsigned int iterations,
	int* num_elements_list,
	unsigned int num_elements_list_size,
	int* max_grid_sizes,
	unsigned int num_max_grid_sizes) 
{
	
	K *h_keys, *h_keys_result;
	
	// Find largest maximum grid size in list of maximum grid sizes
	int max_grid_size = -1;
	for (int i = 0; i < num_max_grid_sizes; i++) {
		if (max_grid_sizes[i] > max_grid_size) {
			max_grid_size = max_grid_sizes[i];
		}
	}

	// find maximum problem size in the list of problem sizes
	unsigned int max_num_elements = 0;
	unsigned int max_num_spine_elements = 0; 
	for (int i = 0; i < num_elements_list_size; i++) {

		RadixSortingEnactor<K, V> sorting_enactor(num_elements_list[i], max_grid_size);
		
		if (sorting_enactor.CanFit()) {
			if (num_elements_list[i] > max_num_elements) {
				max_num_elements = num_elements_list[i];
			}
			unsigned int num_spine_elements = sorting_enactor.SpineElements();
			if (num_spine_elements > max_num_spine_elements) {
				max_num_spine_elements = num_spine_elements;
			}
		}
	}
	
	// Allocate host memory
	h_keys = (K*) malloc(max_num_elements * sizeof(K));
	h_keys_result = (K*) malloc(max_num_elements * sizeof(K));

	// Randomly initialize the keyset on the host
	for (unsigned int j = 0; j < max_num_elements; j++) {
		RandomBits<K>(h_keys[j], g_entropy_reduction);
	}

	// Allocate device memory
	RadixSortStorage<K, V> device_storage;
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.d_keys, max_num_elements * sizeof(K)) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.d_alt_keys, max_num_elements * sizeof(K)));
	if (!KEYS_ONLY) {
		CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.d_values, max_num_elements * sizeof(V)));
		CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.d_alt_values, max_num_elements * sizeof(V)));
	}
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.d_spine, max_num_spine_elements * sizeof(unsigned int)) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.d_from_alt_storage, 2 * sizeof(bool)) );

	// Run combinations of specified problem-sizes & max-grid-sizes
	for (int i = 0; i < num_elements_list_size; i++) {

		if (num_elements_list[i] > max_num_elements) {
			printf("Problem size %d too large\n", num_elements_list[i]);
			continue;
		}

		// Regenerate random keys if specified
		if ((i > 0) && g_regen) {
			for (unsigned int j = 0; j < num_elements_list[i]; j++) {
				RandomBits<K>(h_keys[j], g_entropy_reduction);
			}
		}
		
		for (int j = 0; j < num_max_grid_sizes; j++) {

			// Run a dummy kernel to demarcate the start of this set of iterations in the counter logs
			DummyKernel<<<1,1,0>>>();

			// Run the timing test 
			TimedSort<K, V, KEYS_ONLY>(num_elements_list[i], max_grid_sizes[j], h_keys, h_keys_result, device_storage, iterations);
		}
	}
	
    // cleanup memory
	free(h_keys);
	CUDA_SAFE_CALL(hipFree(device_storage.d_keys));
	CUDA_SAFE_CALL(hipFree(device_storage.d_alt_keys));
	CUDA_SAFE_CALL(hipFree(device_storage.d_spine));
	CUDA_SAFE_CALL(hipFree(device_storage.d_from_alt_storage));
	if (!KEYS_ONLY) {
		CUDA_SAFE_CALL(hipFree(device_storage.d_values));
		CUDA_SAFE_CALL(hipFree(device_storage.d_alt_values));
	}	
}


template<typename K>
void TestSort(
	int value_bytes,
	unsigned int iterations,
	int* num_elements_list,
	unsigned int num_elements_list_size,
	int* max_grid_sizes,
	unsigned int num_max_grid_sizes)
{
	switch (value_bytes) {
	case 0:		// keys only
		TestSort<K, KeysOnlyType, true>(iterations, num_elements_list, num_elements_list_size, max_grid_sizes, num_max_grid_sizes);
		break;
	case 4:		// 32-bit values
		TestSort<K, unsigned int, false>(iterations, num_elements_list, num_elements_list_size, max_grid_sizes, num_max_grid_sizes);
		break;
	case 8:		// 64-bit values
		TestSort<K, unsigned long long, false>(iterations, num_elements_list, num_elements_list_size, max_grid_sizes, num_max_grid_sizes);
		break;
	case 16:	// 128-bit values
		TestSort<K, uint4, false>(iterations, num_elements_list, num_elements_list_size, max_grid_sizes, num_max_grid_sizes);
		break;
	default: 
		fprintf(stderr, "Invalid payload size.  Exiting.\n");
	}
}



/******************************************************************************
 * Main
 ******************************************************************************/

int main( int argc, char** argv) {

	CUT_DEVICE_INIT(argc, argv);

	// presently deterministic
	// srand(time(NULL));	
	srand(0);				

	unsigned int num_elements 				= 512;
    int max_grid_size 						= -1;	// let API determine best grid size
	unsigned int iterations  				= 1;
	char *num_elements_list_filename 		= NULL;
	char *max_grid_sizes_filename 			= NULL;
	int key_bytes							= 4;
	int value_bytes							= 0;
	int* num_elements_list 					= NULL;
	int* max_grid_sizes 					= NULL;
	unsigned int num_elements_list_size;
	unsigned int num_max_grid_sizes;

    //
	// Check command line arguments
    //

    if (cutCheckCmdLineFlag( argc, (const char**) argv, "help")) {
		Usage();
		return 0;
	}

    cutGetCmdLineArgumenti( argc, (const char**) argv, "i", (int*)&iterations);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "key-bytes", (int*)&key_bytes);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "value-bytes", (int*)&value_bytes);
	cutGetCmdLineArgumentstr( argc, (const char**) argv, "n-input", &num_elements_list_filename);
	cutGetCmdLineArgumenti( argc, (const char**) argv, "max-blocks", (int*)&max_grid_size);
	cutGetCmdLineArgumentstr( argc, (const char**) argv, "max-blocks-input", &max_grid_sizes_filename);
	cutGetCmdLineArgumenti( argc, (const char**) argv, "entropy-reduction", (int*)&g_entropy_reduction);
	if (g_verbose2 = cutCheckCmdLineFlag( argc, (const char**) argv, "v2")) {
		g_verbose = true;
	} else {
		g_verbose = cutCheckCmdLineFlag( argc, (const char**) argv, "v");
	}
	g_verify = !cutCheckCmdLineFlag( argc, (const char**) argv, "noverify");
	g_regen = cutCheckCmdLineFlag( argc, (const char**) argv, "regen");
	
	// Attempt to read list of problem sizes to run
	ReadList(
		num_elements_list, 
		num_elements_list_size, 
		num_elements_list_filename, 
		num_elements); 

	// Attempt to read list of max-grid-sizes to run
	ReadList(
		max_grid_sizes, 
		num_max_grid_sizes, 
		max_grid_sizes_filename, 
		max_grid_size); 
	
	// Execute test(s)

	switch (key_bytes) {
	case 1:		// 8-bit keys
		TestSort<unsigned char>(value_bytes, iterations, num_elements_list, num_elements_list_size, max_grid_sizes, num_max_grid_sizes);
		break;
	case 2:		// 16-bit keys
		TestSort<unsigned short>(value_bytes, iterations, num_elements_list, num_elements_list_size, max_grid_sizes, num_max_grid_sizes);
		break;
	case 4:		// 32-bit keys
		TestSort<unsigned int>(value_bytes, iterations, num_elements_list, num_elements_list_size, max_grid_sizes, num_max_grid_sizes);
		break;
	case 8:		// 64-bit keys
		TestSort<unsigned long long>(value_bytes, iterations, num_elements_list, num_elements_list_size, max_grid_sizes, num_max_grid_sizes);
		break;
	default: 
		fprintf(stderr, "Invalid key size.  Exiting.\n");
	}
}



