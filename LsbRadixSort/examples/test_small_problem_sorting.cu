/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * 
 * 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for SRTS Radix Sorting.
 *
 * Useful for demonstrating how to integrate SRTS Radix Sorting into your 
 * application
 ******************************************************************************/

#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <math.h> 
#include <float.h>

// Sorting includes
#include <radixsort_single_grid.cu>
#include <radixsort_early_exit.cu>		
#include <test_utils.cu>				// Utilities and correctness-checking
#include <cutil.h>						// Utilities for commandline parsing

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;



/******************************************************************************
 * Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\ntest_small_problem_sorting [--device=<device index>] [--v] [--i=<num-iterations>] [--n=<num-elements>] [--keys-only]\n"); 
	printf("\n");
	printf("\t--v\tDisplays sorted results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the sorting operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}



/**
 * Uses the small-problem-sorter (single-grid enactor) sort the specified 17-bit sorting
 * problem whose keys is a vector of the specified number of unsigned int elements, 
 * values of unsigned int elements.
 *
 * @param[in] 		num_elements 
 * 		Size in elements of the vector to sort
 * @param[in] 		h_keys 
 * 		Vector of keys to sort 
 * @param[in] 		iterations  
 * 		Number of times to invoke the GPU sorting primitive
 */
void SmallProblemTimedSort(
	unsigned int num_elements, 
	unsigned int *h_keys,
	unsigned int iterations)
{
	printf("Custom single-kernel key-value sort, %d iterations, %d elements", iterations, num_elements);
	
	// Allocate device storage   
	MultiCtaRadixSortStorage<unsigned int, unsigned int> device_storage(num_elements);	
	hipMalloc((void**) &device_storage.d_keys[0], sizeof(unsigned int) * num_elements);
	hipMalloc((void**) &device_storage.d_values[0], sizeof(unsigned int) * num_elements);

	// Create sorting enactor
	SingleGridRadixSortingEnactor<unsigned int, unsigned int> sorting_enactor;

	// Perform a single sorting iteration to allocate memory, prime code caches, etc.
	hipMemcpy(
		device_storage.d_keys[0], 
		h_keys, 
		sizeof(unsigned int) * num_elements, 
		hipMemcpyHostToDevice);		// copy keys
	sorting_enactor.EnactSort<17>(device_storage);

	// Perform the timed number of sorting iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < iterations; i++) {

		RADIXSORT_DEBUG = (i == 0);

		// Move a fresh copy of the problem into device storage
		hipMemcpy(
			device_storage.d_keys[0], 
			h_keys, 
			sizeof(unsigned int) * num_elements, 
			hipMemcpyHostToDevice);		// copy keys

		// Start cuda timing record
		hipEventRecord(start_event, 0);

		// Call the sorting API routine
		sorting_enactor.EnactSort<17>(device_storage);

		// End cuda timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;		
	}

	// Display timing information
	double avg_runtime = elapsed / iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0; 
    printf(", %f GPU ms, %f x10^9 elts/sec\n", 
		avg_runtime,
		throughput);
	
    // Copy out data 
    hipMemcpy(
    	h_keys, 
    	device_storage.d_keys[device_storage.selector], 
    	sizeof(unsigned int) * num_elements, 
    	hipMemcpyDeviceToHost);
    
    // Free allocated memory
    if (device_storage.d_keys[0]) hipFree(device_storage.d_keys[0]);
    if (device_storage.d_keys[1]) hipFree(device_storage.d_keys[1]);
    if (device_storage.d_values[0]) hipFree(device_storage.d_values[0]);
    if (device_storage.d_values[1]) hipFree(device_storage.d_values[1]);

    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
}



/**
 * Uses the large-problem-sorter (early-exit enactor) sort the specified sorting
 * problem whose keys is a vector of the specified number of unsigned int elements, 
 * values of unsigned int elements.
 *
 * @param[in] 		num_elements 
 * 		Size in elements of the vector to sort
 * @param[in] 		h_keys 
 * 		Vector of keys to sort 
 * @param[in] 		iterations  
 * 		Number of times to invoke the GPU sorting primitive
 */
void LargeProblemTimedSort(
	unsigned int num_elements, 
	unsigned int *h_keys,
	unsigned int iterations)
{
	printf("Default key-value sort, %d iterations, %d elements", iterations, num_elements);
	
	// Allocate device storage   
	MultiCtaRadixSortStorage<unsigned int, unsigned int> device_storage(num_elements);	
	hipMalloc((void**) &device_storage.d_keys[0], sizeof(unsigned int) * num_elements);
	hipMalloc((void**) &device_storage.d_values[0], sizeof(unsigned int) * num_elements);

	// Create sorting enactor
	EarlyExitRadixSortingEnactor<unsigned int, unsigned int> sorting_enactor;

	// Perform a single sorting iteration to allocate memory, prime code caches, etc.
	hipMemcpy(
		device_storage.d_keys[0], 
		h_keys, 
		sizeof(unsigned int) * num_elements, 
		hipMemcpyHostToDevice);		// copy keys
	sorting_enactor.EnactSort(device_storage);

	// Perform the timed number of sorting iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < iterations; i++) {

		RADIXSORT_DEBUG = (i == 0);

		// Move a fresh copy of the problem into device storage
		hipMemcpy(
			device_storage.d_keys[0], 
			h_keys, 
			sizeof(unsigned int) * num_elements, 
			hipMemcpyHostToDevice);		// copy keys

		// Start cuda timing record
		hipEventRecord(start_event, 0);

		// Call the sorting API routine
		sorting_enactor.EnactSort(device_storage);

		// End cuda timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;		
	}

	// Display timing information
	double avg_runtime = elapsed / iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0; 
    printf(", %f GPU ms, %f x10^9 elts/sec\n", 
		avg_runtime,
		throughput);
	
    // Copy out data 
    hipMemcpy(
    	h_keys, 
    	device_storage.d_keys[device_storage.selector], 
    	sizeof(unsigned int) * num_elements, 
    	hipMemcpyDeviceToHost);
    
    // Free allocated memory
    if (device_storage.d_keys[0]) hipFree(device_storage.d_keys[0]);
    if (device_storage.d_keys[1]) hipFree(device_storage.d_keys[1]);
    if (device_storage.d_values[0]) hipFree(device_storage.d_values[0]);
    if (device_storage.d_values[1]) hipFree(device_storage.d_values[1]);

    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
}


/**
 * Creates an example sorting problem whose keys is a vector of the specified 
 * number of 17-bit unsigned int elements, values of unsigned int elements, and then 
 * dispatches the problem to the GPU for the given number of iterations, 
 * displaying runtime information.
 *
 * @param[in] 		iterations  
 * 		Number of times to invoke the GPU sorting primitive
 * @param[in] 		num_elements 
 * 		Size in elements of the vector to sort
 * @param[in]		use_small_problem_enactor
 */
void TestSort(
	unsigned int iterations,
	int num_elements, 
	bool use_small_problem_enactor)
{
    // Allocate the sorting problem on the host and fill the keys with random bytes

	unsigned int *h_keys = NULL;
	unsigned int *h_reference_keys = NULL;
	h_keys = (unsigned int*) malloc(num_elements * sizeof(unsigned int));
	h_reference_keys = (unsigned int*) malloc(num_elements * sizeof(unsigned int));

	// Use random bits
	for (unsigned int i = 0; i < num_elements; ++i) {
		RandomBits<unsigned int>(h_keys[i], 0);
		
		// only use 17 effective bits of key data
		h_keys[i] &= (1 << 17) - 1;
		h_reference_keys[i] = h_keys[i];
	}

    // Run the timing test
	if (use_small_problem_enactor) {
		SmallProblemTimedSort(num_elements, h_keys, iterations);
	} else { 
		LargeProblemTimedSort(num_elements, h_keys, iterations);
	}
    
	// Display sorted key data
	if (g_verbose) {
		printf("\n\nKeys:\n");
		for (int i = 0; i < num_elements; i++) {	
			PrintValue<unsigned int>(h_keys[i]);
			printf(", ");
		}
		printf("\n\n");
	}	
	
    // Verify solution
	std::stable_sort(h_reference_keys, h_reference_keys + num_elements);	
	VerifySort<unsigned int>(h_keys, h_reference_keys, num_elements, true);
	printf("\n");
	fflush(stdout);

	// Free our allocated host memory 
	if (h_keys != NULL) free(h_keys);
	if (h_reference_keys != NULL) free(h_reference_keys);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main( int argc, char** argv) {

	CUT_DEVICE_INIT(argc, argv);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

    unsigned int num_elements 					= 1024;
    unsigned int iterations  					= 1;

    //
	// Check command line arguments
    //

    if (cutCheckCmdLineFlag( argc, (const char**) argv, "help")) {
		Usage();
		return 0;
	}

    cutGetCmdLineArgumenti( argc, (const char**) argv, "i", (int*)&iterations);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);
	g_verbose = cutCheckCmdLineFlag( argc, (const char**) argv, "v");

	TestSort(iterations, num_elements, true);	// single-grid enactor (explicit passes)
	TestSort(iterations, num_elements, false); 	// early-exit enactor (dynamic pass detection)
	
	hipDeviceSynchronize();
}



