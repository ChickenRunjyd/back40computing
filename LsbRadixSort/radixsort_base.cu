/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * 
 * 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/



/******************************************************************************
 * Radix Sorting API
 ******************************************************************************/

#pragma once

#include "b40c_kernel_utils.cu"

#include "radixsort_kernel_common.cu"


namespace b40c {

/**
 * Base class for SRTS radix sorting enactors.
 */
template <typename K, typename V, typename Storage>
class BaseRadixSortingEnactor 
{
	
protected:

	/**
	 * Whether or not this instance can be used to sort satellite values
	 */
	static bool KeysOnly() 
	{
		return IsKeysOnly<V>();
	}

protected:

	//Device properties
	const CudaProperties cuda_props;
	
public: 
	
	// Allows display to stdout of sort details
	bool RADIXSORT_DEBUG;
	
protected: 	
	
	/**
	 * Constructor.
	 */
	BaseRadixSortingEnactor(const CudaProperties &props = CudaProperties()) : 
		cuda_props(props), RADIXSORT_DEBUG(false) {}


public:
	

	/**
     * Destructor
     */
    virtual ~BaseRadixSortingEnactor() {}

    
	/**
	 * Enacts a radix sorting operation on the specified device data.
	 *
	 * @return hipSuccess on success, error enumeration otherwise
	 */
	virtual hipError_t EnactSort(Storage &problem_storage) = 0;	
    
};





}// namespace b40c

