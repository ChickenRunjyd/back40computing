/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 * 
 ******************************************************************************/


/******************************************************************************
 * LSB Sorting Base Class
 ******************************************************************************/

#pragma once

#include "b40c_kernel_utils.cu"
#include "radixsort_kernel_common.cu"

namespace b40c {

using namespace lsb_radix_sort;



/**
 * Base class for LSB radix sorting enactors.
 */
template <typename KeyType, typename ValueType, typename StorageType>
class BaseLsbSortEnactor 
{
	
protected:

	/**
	 * Whether or not this instance can be used to sort satellite values
	 */
	static bool KeysOnly() 
	{
		return IsKeysOnly<ValueType>();
	}

protected:

	// Device properties
	const CudaProperties cuda_props;
	
public: 
	
	// Prints sorting debug detail to stdout
	bool DEBUG;
	
protected: 	
	
	/**
	 * Constructor.
	 */
	BaseLsbSortEnactor(const CudaProperties &props = CudaProperties()) : 
		cuda_props(props), DEBUG(false) {}


public:
	

	/**
     * Destructor
     */
    virtual ~BaseLsbSortEnactor() {}

    
	/**
	 * Enacts a radix sorting operation on the specified device data.
	 *
	 * @return hipSuccess on success, error enumeration otherwise
	 */
	virtual hipError_t EnactSort(StorageType &problem_storage) = 0;	
    
};


} // namespace b40c

