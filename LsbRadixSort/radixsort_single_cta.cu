/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * 
 * 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/



#pragma once
#include <hip/hip_runtime.h>



namespace b40c {




}// namespace b40c

