#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for *large-problem* reduction.
 ******************************************************************************/

#include <stdio.h> 

// Reduction includes
#include "reduction_api_granularity.cuh"
#include "reduction_api_enactor_tuned.cuh"

// Test utils
#include "b40c_util.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>


using namespace b40c;
using namespace reduction;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool 	g_verbose 						= false;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;


/******************************************************************************
 * Test wrappers for binary, associative operations
 ******************************************************************************/

template <typename T>
struct Sum
{
	static __host__ __device__ __forceinline__ T Op(const T &a, const T &b)
	{
		return a + b;
	}

	static __host__ __device__ __forceinline__ T Identity()
	{
		return 0;
	}
};

template <typename T>
struct Max
{
	static __host__ __device__ __forceinline__ T Op(const T &a, const T &b)
	{
		return (a > b) ? a : b;
	}

	static __host__ __device__ __forceinline__ T Identity()
	{
		return 0;
	}
};


/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\ntest_reduction [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>] [--sweep]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the reduction operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of bytes to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Timed reduction.  Uses the GPU to copy the specified vector of elements for the given
 * number of iterations, displaying runtime information.
 */
template <
	typename T,
	T BinaryOp(const T&, const T&),
	T Identity()>
double TimedReduction(
	T *h_data,
	T *h_reference,
	size_t num_elements)
{
	T h_dest[1] = {0};
	printf("B40C Reduction: %d iterations, %d elements, ", g_iterations, num_elements);
	
	// Allocate device storage  
	T *d_src, *d_dest;
	if (B40CPerror(hipMalloc((void**) &d_src, sizeof(T) * num_elements),
		"TimedReduction hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);
	if (B40CPerror(hipMalloc((void**) &d_dest, sizeof(T)),
		"TimedReduction hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

	// Create enactor
	ReductionEnactorTuned reduction_enactor;

	// Move a fresh copy of the problem into device storage
	if (B40CPerror(hipMemcpy(d_src, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
		"TimedReduction hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);
	
	// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
	reduction_enactor.template Enact<T, BinaryOp, Identity>(
		d_dest, d_src, num_elements, g_max_ctas);

	// Perform the timed number of iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Start timing record
		hipEventRecord(start_event, 0);

		// Call the reduction API routine
		reduction_enactor.template Enact<T, BinaryOp, Identity>(
			d_dest, d_src, num_elements, g_max_ctas);

		// End timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;		
	}

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
    printf("%f GPU ms, %f x10^9 elts/sec, %f x10^9 B/sec, ",
		avg_runtime, throughput, throughput * sizeof(T));
	
    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);

    // Copy out data
    if (B40CPerror(hipMemcpy(h_dest, d_dest, sizeof(T), hipMemcpyDeviceToHost),
		"TimedReduction hipMemcpy d_dest failed: ", __FILE__, __LINE__)) exit(1);

    // Free allocated memory
    if (d_src) hipFree(d_src);
    if (d_dest) hipFree(d_dest);

	// Flushes any stdio from the GPU
	hipDeviceSynchronize();

	// Display copied data
	if (g_verbose) {
		printf("Reduction: ");
		PrintValue(h_dest[0]);
		printf(", Reference: ");
		PrintValue(h_reference[0]);
		printf("\n\n");
	}

    // Verify solution
	CompareResults(h_dest, h_reference, 1, true);
	printf("\n");
	fflush(stdout);

	return throughput;
}


/**
 * Timed reduction.  Uses the GPU to copy the specified vector of elements for the given
 * number of iterations, displaying runtime information.
 */
template <
	typename T,
	T BinaryOp(const T&, const T&),
	T Identity()>
double TimedThrustReduction(
	T *h_data,
	T *h_reference,
	size_t num_elements)
{
	T h_dest[1] = {0};
	printf("Thrust Reduction: %d iterations, %d elements, ", g_iterations, num_elements);
	
	// Allocate device storage  
	T *d_src, *d_dest;
	if (B40CPerror(hipMalloc((void**) &d_src, sizeof(T) * num_elements),
		"TimedReduction hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);
	if (B40CPerror(hipMalloc((void**) &d_dest, sizeof(T)),
		"TimedReduction hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

	// Move a fresh copy of the problem into device storage
	if (B40CPerror(hipMemcpy(d_src, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
		"TimedReduction hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);
	
	// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
	thrust::device_ptr<T> dev_ptr(d_src);		
	h_dest[0] = thrust::reduce(dev_ptr, dev_ptr + num_elements, (T) 0, thrust::plus<T>());
	
	// Perform the timed number of iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Start timing record
		hipEventRecord(start_event, 0);

		h_dest[0] = thrust::reduce(dev_ptr, dev_ptr + num_elements, (T) 0, thrust::plus<T>());
		
		// End timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;		
	}

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
    printf("%f GPU ms, %f x10^9 elts/sec, %f x10^9 B/sec, ",
		avg_runtime, throughput, throughput * sizeof(T));
	
    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);

    // Free allocated memory
    if (d_src) hipFree(d_src);
    if (d_dest) hipFree(d_dest);

	// Flushes any stdio from the GPU
	hipDeviceSynchronize();

	// Display copied data
	if (g_verbose) {
		printf("Reduction: ");
		PrintValue(h_dest[0]);
		printf(", Reference: ");
		PrintValue(h_reference[0]);
		printf("\n\n");
	}

    // Verify solution
	CompareResults(h_dest, h_reference, 1, true);
	printf("\n");
	fflush(stdout);

	return throughput;
}





/**
 * Creates an example reduction problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	T BinaryOp(const T&, const T&),
	T Identity()>
void TestReduction(size_t num_elements)
{
    // Allocate the reduction problem on the host and fill the keys with random bytes

	T *h_data 			= (T*) malloc(num_elements * sizeof(T));
	T *h_reference 		= (T*) malloc(sizeof(T));

	if ((h_data == NULL) || (h_reference == NULL)){
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	// Identity
	h_reference[0] = Identity();

	for (size_t i = 0; i < num_elements; ++i) {
		// RandomBits<T>(h_data[i], 0);
		h_data[i] = i;
		h_reference[0] = BinaryOp(h_reference[0], h_data[i]);
	}

	//
    // Run the timing test(s)
	//
	double b40c = TimedReduction<T, BinaryOp, Identity>(h_data, h_reference, num_elements);
	double thrust = TimedThrustReduction<T, BinaryOp, Identity>(h_data, h_reference, num_elements);
	printf("B40C speedup: %.2f\n", b40c/thrust);
	

	// Free our allocated host memory 
	if (h_data) free(h_data);
    if (h_reference) free(h_reference);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

    //
	// Check command line arguments
    //

	size_t num_elements = 1024;

    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}

    bool sweep = args.CheckCmdLineFlag("sweep");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	// Execute test(s), optionally sweeping problem size downward
	{
		printf("\n-- UNSIGNED CHAR ----------------------------------------------\n");
		typedef unsigned char T;
		typedef Sum<T> BinaryOp;
		TestReduction<T, BinaryOp::Op, BinaryOp::Identity>(num_elements * 4);
	}
	{
		printf("\n-- UNSIGNED SHORT ----------------------------------------------\n");
		typedef unsigned short T;
		typedef Sum<T> BinaryOp;
		TestReduction<T, BinaryOp::Op, BinaryOp::Identity>(num_elements * 2);
	}
	{
		printf("\n-- UNSIGNED INT -----------------------------------------------\n");
		typedef unsigned int T;
		typedef Sum<T> BinaryOp;
		TestReduction<T, BinaryOp::Op, BinaryOp::Identity>(num_elements);
	}
	{
		printf("\n-- UNSIGNED LONG LONG -----------------------------------------\n");
		typedef unsigned long long T;
		typedef Sum<T> BinaryOp;
		TestReduction<T, BinaryOp::Op, BinaryOp::Identity>(num_elements / 2);
	}
	
	
	
/*	
    size_t orig_num_elements = num_elements;
    do {

    	TestReduction<T, BinaryOp::Op, BinaryOp::Identity>(num_elements);
    	num_elements -= 4096;

    } while (sweep && (num_elements < orig_num_elements ));
*/    

	return 0;
}



