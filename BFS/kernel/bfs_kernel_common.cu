#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 * 
 ******************************************************************************/

/******************************************************************************
 * Common types and kernel routines for B40C BFS kernels  
 ******************************************************************************/

#pragma once

#include <b40c_kernel_utils.cu>
#include <b40c_vector_types.cu>

namespace b40c {


/******************************************************************************
 * BFS Algorithm and Granularity Configuration 
 ******************************************************************************/

/**
 * CTA size in threads
 */
#define B40C_BFS_SG_LOG_THREADS								(7)			// 128 threads								
#define B40C_BFS_SG_THREADS									(1 << B40C_BFS_SG_LOG_THREADS)	

/**
 * Enumeration of parallel BFS algorithm strategies
 */
enum BfsStrategy {
	
	/**
	 * Contract-then-Expand
	 * 
	 * At each iteration, the frontier queue is comprised of "unvisited-edges", 
	 * i.e. the concatenation of the adjacency lists belonging to the nodes 
	 * that were discovered from the previous iteration. (These unvisited-edges 
	 * are simply the incident node IDs.)  The algorithm discards the edges 
	 * leading to previously-visited nodes, and then expands the edge-lists of 
	 * the remaining (newly-discovered) nodes into the frontier queue for the 
	 * next iteration. As the frontier is streamed through the SMs for each BFS 
	 * iteration, the kernel operates by:
	 * 
	 *  (1) Streaming in tiles of the frontier queue and contracting the unvisited-
	 *      edges by:
	 *        (i)  Removing incident nodes that were discovered by previous 
	 *             iterations
	 *        (ii) A heuristic for removing duplicate incident nodes��. 
	 *         
	 *      The remaining incident nodes are marked as being discovered at this 
	 *      iteration. 
	 *       
	 *  (2) Expanding the newly discovered nodes into their adjacency lists and
	 *      enqueuing these adjacency lists into the outgoing frontier for 
	 *      processing by the next iteration. 
	 */
	CONTRACT_EXPAND,
	
	/**
	 * Expand-then-Contract 
	 * 
	 * At each iteration, the frontier queue is comprised of "discovered nodes" 
	 * from the previous iteration.  The algorithm expands these nodes into 
	 * their edge-lists.  The edges leading to previously-visited nodes are 
	 * discarded.  Then the remaining (newly-discovered) nodes are enqueued 
	 * into the frontier queue for the next iteration. As the frontier is 
	 * streamed through the SMs for each BFS iteration, the kernel operates by:
	 * 
	 *  (1) Streaming in tiles of the frontier queue and expanding those nodes 
	 *      into their adjacency lists into shared-memory scratch space.
	 *         
	 *  (2) Contracting these "unvisited edges" in shared-memory scratch by:
	 *        (i)  Removing incident nodes that were discovered by previous 
	 *             iterations
	 *        (ii) A heuristic for removing duplicate incident nodes��. 
	 *         
	 *      The remaining incident nodes are marked as being discovered at this 
	 *      iteration, and enqueued into the outgoing frontier for processing by 
	 *      the next iteration.
	 */
	EXPAND_CONTRACT
	
	/**
	 * Footnotes:
	 * 
	 *   �� Frontier duplicates exist when a node is neighbor to multiple nodes 
	 *      discovered by the previous iteration.  Although the operation of the 
	 *      algorithm is correct regardless of the number of times a node is 
	 *      discovered within a given iteration, duplicate-removal can drastically 
	 *      reduce the overall work performed.  When the same node is discovered
	 *      concurrently within a given iteration, its entire adjacency list will 
	 *      be duplicated in the next iteration's frontier.  Duplicate-removal is 
	 *      particularly effective for lattice-like graphs: nodes are often 
	 *      discoverable at a given iteration via multiple indicent edges.  
	 */
};


/******************************************************************************
 * BFS Kernel Subroutines 
 ******************************************************************************/

/**
 * Perform a local prefix sum to rank the specified partial_reductions
 * vector, storing the results in the corresponding local_ranks vector.
 * Also performs an atomic-increment at the d_queue_length address with the 
 * aggregate, storing the previous value in s_enqueue_offset.  Returns the 
 * aggregate.  
 * 
 * Needs a subsequent syncthreads for safety of further scratch_pool usage
 * 
 * Currently only supports RAKING_THREADS = B40C_WARP_THREADS.
 * Currently only supports LOADS_PER_TILE = 1.
 */
template <int LOAD_VEC_SIZE, int ELEMENTS_PER_SEGMENT>
__device__ __forceinline__ 
int LocalScanWithAtomicReservation(
	int *base_partial,
	int *raking_segment,
	int warpscan[2][B40C_WARP_THREADS],
	int partial_reductions[LOAD_VEC_SIZE],
	int local_ranks[LOAD_VEC_SIZE],
	int *d_queue_length,
	int &s_enqueue_offset)
{
	// Reduce in registers, placing the result into our smem cell for raking
	base_partial[0] = SerialReduce<int, LOAD_VEC_SIZE>(partial_reductions);

	__syncthreads();

	// Rake-reduce, warpscan, and rake-scan.
	if (threadIdx.x < B40C_WARP_THREADS) {

		// Serial reduce (rake) in smem
		int raked_reduction = SerialReduce<int, ELEMENTS_PER_SEGMENT>(raking_segment);

		// Warpscan
		int seed = WarpScan<B40C_WARP_THREADS, false>(warpscan, raked_reduction);
		
		// Atomic-increment the global counter with our cycle's allocation
		if (threadIdx.x == 0) {
			s_enqueue_offset = atomicAdd(d_queue_length, warpscan[1][B40C_WARP_THREADS - 1]);
		}
		
		// Serial scan (rake) in smem
		SerialScan<int, ELEMENTS_PER_SEGMENT>(raking_segment, seed);
	}

	__syncthreads();

	SerialScan<int, LOAD_VEC_SIZE>(partial_reductions, local_ranks, base_partial[0]);
	
	return warpscan[1][B40C_WARP_THREADS - 1];
}


/**
 * Loads a single IndexType from the specified offset into node_id
 * if in range, otherwise node_id is assigned -1 instead  
 */
template <typename IndexType, int SCRATCH_SPACE, CacheModifier LIST_MODIFIER>
__device__ __forceinline__
void GuardedSingletonLoad(
	IndexType &node_id,			
	int &hash,
	IndexType *scratch_pool,
	IndexType *node_id_list,
	int load_offset,
	int out_of_bounds)							 
{
	if (load_offset < out_of_bounds) {
		ModifiedLoad<IndexType, LIST_MODIFIER>::Ld(node_id, node_id_list, load_offset);
		hash = node_id % SCRATCH_SPACE;
		scratch_pool[hash] = node_id;
	} else {
		node_id = -1;
		hash = SCRATCH_SPACE - 1;
	}
}


/**
 * Uses vector-loads to read a tile of node-IDs from the node_id_list 
 * reference, optionally conditional on bounds-checking.  Performs a 
 * conservative culling of duplicate node-IDs based upon a linear hashing of 
 * the node-IDs.  The corresponding duplicate flag is set to true for a given 
 * node-ID if it can be verified that some other thread will set its own 
 * duplicate flag false for the same node-ID, false otherwise. 
 * 
 * Needs a subsequent syncthreads for safety of further scratch_pool usage
 */
template <
	typename IndexType, 
	int SCRATCH_SPACE, 
	int LOAD_VEC_SIZE, 
	CacheModifier LIST_MODIFIER,
	bool UNGUARDED_IO>
__device__ __forceinline__
void LoadAndCullDuplicates(
	IndexType node_id[LOAD_VEC_SIZE],		// out param
	bool duplicate[LOAD_VEC_SIZE],			// out param
	IndexType *node_id_list,
	int out_of_bounds,							 
	IndexType *scratch_pool)						 
{
	// Hash offset for each node-ID
	int hash[LOAD_VEC_SIZE];		

	// Initially label everything as a duplicate
	#pragma unroll
	for (int COMPONENT = 0; COMPONENT < LOAD_VEC_SIZE; COMPONENT++) {
		duplicate[COMPONENT] = true;				
	}

	// Load node-IDs
	if (UNGUARDED_IO) {
		
		// Use a built-in, vector-typed alias to load straight into node_id array
		typedef typename VecType<IndexType, LOAD_VEC_SIZE>::Type BuiltinVec; 		

		BuiltinVec *node_id_list_vec = (BuiltinVec *) node_id_list;
		BuiltinVec *built_in_alias = (BuiltinVec *) node_id;
		ModifiedLoad<BuiltinVec, LIST_MODIFIER>::Ld(*built_in_alias, node_id_list_vec, threadIdx.x);

		// Hash the node-IDs into smem scratch
		#pragma unroll
		for (int COMPONENT = 0; COMPONENT < LOAD_VEC_SIZE; COMPONENT++) {
			hash[COMPONENT] = node_id[COMPONENT] % SCRATCH_SPACE;
			scratch_pool[hash[COMPONENT]] = node_id[COMPONENT];
		}
		
	} else {
		
		// N.B.: Wish we could unroll here, but can't use inlined ASM instructions
		// in a pragma-unroll.

		if (LOAD_VEC_SIZE > 0) {
			GuardedSingletonLoad<IndexType, SCRATCH_SPACE, LIST_MODIFIER>(
				node_id[0], hash[0], scratch_pool, node_id_list, (B40C_BFS_SG_THREADS * 0) + threadIdx.x, out_of_bounds);
		}
		if (LOAD_VEC_SIZE > 1) {
			GuardedSingletonLoad<IndexType, SCRATCH_SPACE, LIST_MODIFIER>(
				node_id[1], hash[1], scratch_pool, node_id_list, (B40C_BFS_SG_THREADS * 1) + threadIdx.x, out_of_bounds);
		}
		if (LOAD_VEC_SIZE > 2) {
			GuardedSingletonLoad<IndexType, SCRATCH_SPACE, LIST_MODIFIER>(
				node_id[2], hash[2], scratch_pool, node_id_list, (B40C_BFS_SG_THREADS * 2) + threadIdx.x, out_of_bounds);
		}
		if (LOAD_VEC_SIZE > 3) {
			GuardedSingletonLoad<IndexType, SCRATCH_SPACE, LIST_MODIFIER>(
				node_id[3], hash[3], scratch_pool, node_id_list, (B40C_BFS_SG_THREADS * 3) + threadIdx.x, out_of_bounds);
		}
	}
	
	__syncthreads();
	
	// Retrieve what node-IDs "won" at those locations
	int hashed_node_id[LOAD_VEC_SIZE];	
	
	#pragma unroll
	for (int COMPONENT = 0; COMPONENT < LOAD_VEC_SIZE; COMPONENT++) {

		hashed_node_id[COMPONENT] = scratch_pool[hash[COMPONENT]];
		if (hashed_node_id[COMPONENT] != node_id[COMPONENT]) {

			// A different node beat us to this hash cell; we must assume 
			// that we may not be a duplicate
			duplicate[COMPONENT] = false;
		}
	}
	
	__syncthreads();
	
	// For the winners, hash in thread-IDs to select one of the threads
	#pragma unroll
	for (int COMPONENT = 0; COMPONENT < LOAD_VEC_SIZE; COMPONENT++) {
		if (hashed_node_id[COMPONENT] == node_id[COMPONENT]) {
			scratch_pool[hash[COMPONENT]] = threadIdx.x;
		}
	}
	
	__syncthreads();
	
	// See if our thread won out amongst everyone with similar node-IDs 
	#pragma unroll
	for (int COMPONENT = 0; COMPONENT < LOAD_VEC_SIZE; COMPONENT++) {
		if (hashed_node_id[COMPONENT] == node_id[COMPONENT]) {
			if (scratch_pool[hash[COMPONENT]] == threadIdx.x) {

				// We are an authoritative (non-duplicate) thread for this node-ID
				duplicate[COMPONENT] = false;
			}
		}
	}
}
	

/**
 * Inspects an incident node-ID to see if it's been visited already.  If not,
 * we mark its discovery in d_source_dist at this iteration, returning 
 * the length and offset of its neighbor row.  If not, we return zero as the 
 * length of its neighbor row.
 */
template <
	typename IndexType, 
	int SCRATCH_SPACE, 
	CacheModifier SOURCE_DIST_MODIFIER,
	CacheModifier ROW_OFFSETS_MODIFIER,
	CacheModifier MISALIGNED_ROW_OFFSETS_MODIFIER>
__device__ __forceinline__
void InspectAndUpdate(
	IndexType node_id,		
	int &row_offset,				// out param
	int &row_length,				// out param
	IndexType *d_source_dist,
	IndexType *d_row_offsets,
	IndexType iteration)
{
	// Load source distance of node
	int source_dist;
	ModifiedLoad<int, SOURCE_DIST_MODIFIER>::Ld(source_dist, d_source_dist, node_id);

	if (source_dist == -1) {
		// Node is previously unvisited.  Load neighbor row range from d_row_offsets
		int2 row_range;
		if (node_id & 1) {
			// Misaligned
			ModifiedLoad<int, MISALIGNED_ROW_OFFSETS_MODIFIER>::Ld(row_range.x, d_row_offsets, node_id);
			ModifiedLoad<int, MISALIGNED_ROW_OFFSETS_MODIFIER>::Ld(row_range.y, d_row_offsets, node_id + 1);
		} else {
			// Aligned
			int2* d_row_offsets_v2 = reinterpret_cast<int2*>(d_row_offsets + node_id);
			ModifiedLoad<int2, ROW_OFFSETS_MODIFIER>::Ld(row_range, d_row_offsets_v2, 0);
		}
		// Compute row offset and length
		row_offset = row_range.x;
		row_length = row_range.y - row_range.x;

		// Update distance with current iteration
		d_source_dist[node_id] = iteration;
	}
}


/**
 * Process a single tile of work from the current incoming frontier queue
 */
template <
	typename IndexType,
	int PARTIALS_PER_SEG, 
	int SCRATCH_SPACE, 
	int LOAD_VEC_SIZE,
	CacheModifier QUEUE_MODIFIER,
	CacheModifier COLUMN_INDICES_MODIFIER,
	CacheModifier SOURCE_DIST_MODIFIER,
	CacheModifier ROW_OFFSETS_MODIFIER,
	CacheModifier MISALIGNED_ROW_OFFSETS_MODIFIER,
	bool UNGUARDED_IO>
__device__ __forceinline__ 
void BfsTile(
	IndexType iteration,
	IndexType *scratch_pool,
	int *base_partial,
	int *raking_segment,
	int warpscan[2][B40C_WARP_THREADS],
	IndexType *d_in_queue, 
	IndexType *d_out_queue,
	IndexType *d_column_indices,
	IndexType *d_row_offsets,
	IndexType *d_source_dist,
	int *d_queue_length,
	int &s_enqueue_offset,
	int cta_out_of_bounds)
{
	IndexType dequeued_node_id[LOAD_VEC_SIZE];	// Incoming node-IDs to process for this tile
	bool duplicate[LOAD_VEC_SIZE];				// Whether or not the node-ID is a guaranteed duplicate
	IndexType row_offset[LOAD_VEC_SIZE];		// The offset into column_indices for retrieving the neighbor list
	IndexType row_length[LOAD_VEC_SIZE];		// Number of adjacent neighbors
	int local_rank[LOAD_VEC_SIZE];				// Prefix sum of row-lengths, i.e., local rank for where to plop down neighbor list into scratch 
	int row_progress[LOAD_VEC_SIZE];			// Iterator for the neighbor list
	int cta_progress = 0;						// Progress of the CTA as a whole towards writing out all neighbors to the outgoing queue

	// Initialize neighbor-row-length (and progress through that row) to zero.
	#pragma unroll
	for (int COMPONENT = 0; COMPONENT < LOAD_VEC_SIZE; COMPONENT++) {
		row_length[COMPONENT] = 0;
		row_progress[COMPONENT] = 0;
	}
	
	//
	// Dequeue a tile of incident node-IDs to explore and use a heuristic for 
	// culling duplicates
	//

	LoadAndCullDuplicates<IndexType, SCRATCH_SPACE, LOAD_VEC_SIZE, QUEUE_MODIFIER, UNGUARDED_IO>(
		dequeued_node_id,			// out param
		duplicate,					// out param
		d_in_queue,
		cta_out_of_bounds,							 
		scratch_pool);	
	
	__syncthreads();

	//
	// Inspect visitation status of incident node-IDs, acquiring row offsets 
	// and lengths for previously-undiscovered node-IDs
	//
	// N.B.: Wish we could unroll here, but can't use inlined ASM instructions
	// in a pragma-unroll.
	//

	if (LOAD_VEC_SIZE > 0) {
		if ((!duplicate[0]) && (UNGUARDED_IO || (dequeued_node_id[0] != -1))) {
			InspectAndUpdate<IndexType, SCRATCH_SPACE, SOURCE_DIST_MODIFIER, ROW_OFFSETS_MODIFIER, MISALIGNED_ROW_OFFSETS_MODIFIER>(
				dequeued_node_id[0], row_offset[0], row_length[0], d_source_dist, d_row_offsets, iteration);
		}
	}
	if (LOAD_VEC_SIZE > 1) {
		if ((!duplicate[1]) && (UNGUARDED_IO || (dequeued_node_id[1] != -1))) {
			InspectAndUpdate<IndexType, SCRATCH_SPACE, SOURCE_DIST_MODIFIER, ROW_OFFSETS_MODIFIER, MISALIGNED_ROW_OFFSETS_MODIFIER>(
				dequeued_node_id[1], row_offset[1], row_length[1], d_source_dist, d_row_offsets, iteration);
		}
	}
	if (LOAD_VEC_SIZE > 2) {
		if ((!duplicate[2]) && (UNGUARDED_IO || (dequeued_node_id[2] != -1))) {
			InspectAndUpdate<IndexType, SCRATCH_SPACE, SOURCE_DIST_MODIFIER, ROW_OFFSETS_MODIFIER, MISALIGNED_ROW_OFFSETS_MODIFIER>(
				dequeued_node_id[2], row_offset[2], row_length[2], d_source_dist, d_row_offsets, iteration);
		}
	}
	if (LOAD_VEC_SIZE > 3) {
		if ((!duplicate[3]) && (UNGUARDED_IO || (dequeued_node_id[3] != -1))) {
			InspectAndUpdate<IndexType, SCRATCH_SPACE, SOURCE_DIST_MODIFIER, ROW_OFFSETS_MODIFIER, MISALIGNED_ROW_OFFSETS_MODIFIER>(
				dequeued_node_id[3], row_offset[3], row_length[3], d_source_dist, d_row_offsets, iteration);
		}
	}
	

	//
	// Perform local scan of neighbor-counts and reserve a spot for them in 
	// the outgoing queue at s_enqueue_offset
	//

	int enqueue_count = LocalScanWithAtomicReservation<LOAD_VEC_SIZE, PARTIALS_PER_SEG>(
		base_partial, raking_segment, warpscan, row_length, local_rank, d_queue_length, s_enqueue_offset);

	__syncthreads();

	
	//
	// Enqueue the adjacency lists of unvisited node-IDs by repeatedly 
	// constructing a set of gather-offsets in the scratch space, and then 
	// having the entire CTA use them to copy adjacency lists from 
	// column_indices to the outgoing frontier queue.
	//

	while (cta_progress < enqueue_count) {
	
		//
		// Fill the scratch space with gather-offsets for neighbor-lists.  Wish we could 
		// pragma unroll here, but we can't do that with inner loops
		// 

		if (LOAD_VEC_SIZE > 0) {
			const int COMPONENT = 0;
			// Attempt to make futher progress on neighbor list
			int scratch_offset = local_rank[COMPONENT] + row_progress[COMPONENT] - cta_progress;
			while ((row_progress[COMPONENT] < row_length[COMPONENT]) && (scratch_offset < SCRATCH_SPACE)) {
				
				// Put a gather offset into the scratch space
				scratch_pool[scratch_offset] = row_offset[COMPONENT] + row_progress[COMPONENT];
				row_progress[COMPONENT]++;
				scratch_offset++;
			}
		}
		if (LOAD_VEC_SIZE > 1) {
			const int COMPONENT = 1;
			// Attempt to make futher progress on neighbor list
			int scratch_offset = local_rank[COMPONENT] + row_progress[COMPONENT] - cta_progress;
			while ((row_progress[COMPONENT] < row_length[COMPONENT]) && (scratch_offset < SCRATCH_SPACE)) {
				
				// Put a gather offset into the scratch space
				scratch_pool[scratch_offset] = row_offset[COMPONENT] + row_progress[COMPONENT];
				row_progress[COMPONENT]++;
				scratch_offset++;
			}
		}
		if (LOAD_VEC_SIZE > 2) {
			const int COMPONENT = 2;
			// Attempt to make futher progress on neighbor list
			int scratch_offset = local_rank[COMPONENT] + row_progress[COMPONENT] - cta_progress;
			while ((row_progress[COMPONENT] < row_length[COMPONENT]) && (scratch_offset < SCRATCH_SPACE)) {
				
				// Put a gather offset into the scratch space
				scratch_pool[scratch_offset] = row_offset[COMPONENT] + row_progress[COMPONENT];
				row_progress[COMPONENT]++;
				scratch_offset++;
			}
		}
		if (LOAD_VEC_SIZE > 3) {
			const int COMPONENT = 3;
			// Attempt to make futher progress on neighbor list
			int scratch_offset = local_rank[COMPONENT] + row_progress[COMPONENT] - cta_progress;
			while ((row_progress[COMPONENT] < row_length[COMPONENT]) && (scratch_offset < SCRATCH_SPACE)) {
				
				// Put a gather offset into the scratch space
				scratch_pool[scratch_offset] = row_offset[COMPONENT] + row_progress[COMPONENT];
				row_progress[COMPONENT]++;
				scratch_offset++;
			}
		}
		
		__syncthreads();
		
		//
		// Copy adjacency lists from column-indices to outgoing queue
		//

		int remainder = B40C_MIN(SCRATCH_SPACE, enqueue_count - cta_progress);
		for (int scratch_offset = threadIdx.x; scratch_offset < remainder; scratch_offset += B40C_BFS_SG_THREADS) {

			// Gather
			int node_id;
			ModifiedLoad<IndexType, COLUMN_INDICES_MODIFIER>::Ld(
				node_id, d_column_indices, scratch_pool[scratch_offset]);
			
			// Scatter
			d_out_queue[s_enqueue_offset + cta_progress + scratch_offset] = node_id;
		}

		cta_progress += SCRATCH_SPACE;
		
		__syncthreads();
	}
}


/**
 * Processes a BFS iteration through the current incoming frontier queue
 */
template <
	BfsStrategy STRATEGY,
	typename IndexType,
	int TILE_ELEMENTS,
	int PARTIALS_PER_SEG, 
	int SCRATCH_SPACE, 
	int LOAD_VEC_SIZE,
	CacheModifier QUEUE_MODIFIER,
	CacheModifier COLUMN_INDICES_MODIFIER,
	CacheModifier SOURCE_DIST_MODIFIER,
	CacheModifier ROW_OFFSETS_MODIFIER,
	CacheModifier MISALIGNED_ROW_OFFSETS_MODIFIER>
__device__ __forceinline__ 
void BfsIteration(
	IndexType iteration,
	IndexType *scratch_pool,
	int *base_partial,
	int *raking_segment,
	int warpscan[2][B40C_WARP_THREADS],
	IndexType *d_in_queue, 
	IndexType *d_out_queue,
	IndexType *d_column_indices,
	IndexType *d_row_offsets,
	IndexType *d_source_dist,
	int *d_queue_length,
	int &s_enqueue_offset,
	int cta_offset, 
	int cta_extra_elements,
	int cta_out_of_bounds)
{
	// Process all of our full-sized tiles (unguarded loads)
	while (cta_offset <= cta_out_of_bounds - TILE_ELEMENTS) {

		BfsTile<IndexType, PARTIALS_PER_SEG, SCRATCH_SPACE, LOAD_VEC_SIZE, 
				QUEUE_MODIFIER, COLUMN_INDICES_MODIFIER, SOURCE_DIST_MODIFIER, 
				ROW_OFFSETS_MODIFIER, MISALIGNED_ROW_OFFSETS_MODIFIER, true>( 
			iteration,
			scratch_pool,
			base_partial,
			raking_segment,
			warpscan,
			d_in_queue + cta_offset, 
			d_out_queue,
			d_column_indices,
			d_row_offsets,
			d_source_dist,
			d_queue_length,
			s_enqueue_offset,
			TILE_ELEMENTS);

		cta_offset += TILE_ELEMENTS;
	}

	// Cleanup any remainder elements (guarded_loads)
	if (cta_extra_elements) {
		
		BfsTile<IndexType, PARTIALS_PER_SEG, SCRATCH_SPACE, LOAD_VEC_SIZE, 
				QUEUE_MODIFIER, COLUMN_INDICES_MODIFIER, SOURCE_DIST_MODIFIER, 
				ROW_OFFSETS_MODIFIER, MISALIGNED_ROW_OFFSETS_MODIFIER, false>( 
			iteration,
			scratch_pool,
			base_partial,
			raking_segment,
			warpscan,
			d_in_queue + cta_offset, 
			d_out_queue,
			d_column_indices,
			d_row_offsets,
			d_source_dist,
			d_queue_length,
			s_enqueue_offset,
			cta_extra_elements); 
	}
}


} // b40c namespace


