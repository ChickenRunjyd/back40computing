#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Tuning tool for establishing optimal memcopy granularity configuration types
 ******************************************************************************/

#include <stdio.h> 

// Memcopy includes
#include "memcopy_api_granularity.cuh"
#include "memcopy_api_enactor.cuh"

// Test utils
#include "b40c_util.h"

using namespace b40c;
using namespace memcopy;

/******************************************************************************
 * Defines, constants, globals, and utility types
 ******************************************************************************/

bool g_verbose;
int g_max_ctas = 0;
int g_iterations = 0;


template <typename T>
struct Detail
{
	MemcopyEnactor<> memcopy_enactor;
	T *d_dest;
	T *d_src;
	size_t num_elements;

	Detail(size_t num_elements) : d_dest(NULL), d_src(NULL), num_elements(num_elements) {}
};


/******************************************************************************
 * Utility routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntune_memcopy_large [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the memcopy operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Timed memcopy for a specific granularity configuration type
 */
template <typename Config>
void TimedMemcopy(Detail<typename Config::T> &detail)
{
	Config::Print();

	// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
	detail.memcopy_enactor.DEBUG = g_verbose;
	detail.memcopy_enactor.template Enact<Config>(detail.d_dest, detail.d_src, detail.num_elements, g_max_ctas);
	detail.memcopy_enactor.DEBUG = false;

	// Perform the timed number of iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Start cuda timing record
		hipEventRecord(start_event, 0);

		// Call the memcopy API routine
		detail.memcopy_enactor.template Enact<Config>(detail.d_dest, detail.d_src, detail.num_elements, g_max_ctas);

		// End cuda timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;
	}

	// Flushes any stdio from the GPU
	hipDeviceSynchronize();

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput = ((double) detail.num_elements) / avg_runtime / 1000.0 / 1000.0;
    printf(", %f, %f, %f\n",
		avg_runtime, throughput, 2 * throughput * sizeof(Config::T));
    fflush(stdout);

    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
}


/******************************************************************************
 * Kernel configuration sweep types
 ******************************************************************************/

enum Ranges
{
	MIN_LOG_THREADS 			= 5,
	MAX_LOG_THREADS 			= 10 + 1,

	MIN_LOG_LOAD_VEC_SIZE 		= 0,
	MAX_LOG_LOAD_VEC_SIZE 		= 2 + 1,

	MIN_LOG_LOADS_PER_TILE 		= 0,
	MAX_LOG_LOADS_PER_TILE 		= 2 + 1,

	MIN_CACHE_MODIFIER 			= NONE,
	MAX_CACHE_MODIFIER 			= LIMIT,

	MIN_WORK_STEALING 			= 0,
	MAX_WORK_STEALING 			= 1 + 1
};


template <int CUDA_ARCH, typename T>
struct SweepConfig
{
	// Next WORK_STEALING
	template <int LOG_THREADS, int LOG_LOAD_VEC_SIZE, int LOG_LOADS_PER_TILE, int CACHE_MODIFIER, int WORK_STEALING>
	struct Iterate
	{
		static void Invoke(Detail<T> &detail)
		{
			// Invoke this config
			const int CTA_OCCUPANCY = B40C_MIN(B40C_SM_CTAS(CUDA_ARCH), (B40C_SM_THREADS(CUDA_ARCH)) >> LOG_THREADS);
			typedef MemcopyConfig<typename T, CTA_OCCUPANCY, LOG_THREADS, LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE, (CacheModifier) CACHE_MODIFIER, WORK_STEALING> Config;
			TimedMemcopy<Config>(detail);

			// Next WORK_STEALING
			Iterate<LOG_THREADS, LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE, CACHE_MODIFIER, WORK_STEALING + 1>::Invoke(detail);
		}
	};

	// Last WORK_STEALING, next CACHE_MODIFIER
	template <int LOG_THREADS, int LOG_LOAD_VEC_SIZE, int LOG_LOADS_PER_TILE, int CACHE_MODIFIER>
	struct Iterate<LOG_THREADS, LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE, CACHE_MODIFIER, MAX_WORK_STEALING>
	{
		static void Invoke(Detail<T> &detail)
		{
			// Next CACHE_MODIFIER (reset WORK_STEALING)
			Iterate<LOG_THREADS, LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE, CACHE_MODIFIER + 1, MIN_WORK_STEALING>::Invoke(detail);
		}
	};

	// Last CACHE_MODIFIER, next LOG_LOADS_PER_TILE
	template <int LOG_THREADS, int LOG_LOAD_VEC_SIZE, int LOG_LOADS_PER_TILE, int WORK_STEALING>
	struct Iterate<LOG_THREADS, LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE, MAX_CACHE_MODIFIER, WORK_STEALING>
	{
		static void Invoke(Detail<T> &detail)
		{
			// Next LOG_LOADS_PER_TILE (reset CACHE_MODIFIER)
			Iterate<LOG_THREADS, LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE + 1, MIN_CACHE_MODIFIER, MIN_WORK_STEALING>::Invoke(detail);
		}
	};

	// Last LOG_LOADS_PER_TILE, next LOG_LOAD_VEC_SIZE
	template <int LOG_THREADS, int LOG_LOAD_VEC_SIZE, int CACHE_MODIFIER, int WORK_STEALING>
	struct Iterate<LOG_THREADS, LOG_LOAD_VEC_SIZE, MAX_LOG_LOADS_PER_TILE, CACHE_MODIFIER, WORK_STEALING>
	{
		static void Invoke(Detail<T> &detail)
		{
			// Next LOG_LOAD_VEC_SIZE (reset LOG_LOADS_PER_TILE)
			Iterate<LOG_THREADS, LOG_LOAD_VEC_SIZE + 1, MIN_LOG_LOADS_PER_TILE, MIN_CACHE_MODIFIER, MIN_WORK_STEALING>::Invoke(detail);
		}
	};

	// Last LOG_LOAD_VEC_SIZE, next LOG_THREADS
	template <int LOG_THREADS, int LOG_LOADS_PER_TILE, int CACHE_MODIFIER, int WORK_STEALING>
	struct Iterate<LOG_THREADS, MAX_LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE, CACHE_MODIFIER, WORK_STEALING>
	{
		static void Invoke(Detail<T> &detail)
		{
			// Next LOG_THREADS (reset LOG_LOAD_VEC_SIZE)
			Iterate<LOG_THREADS + 1, MIN_LOG_LOAD_VEC_SIZE, MIN_LOG_LOADS_PER_TILE, MIN_CACHE_MODIFIER, MIN_WORK_STEALING>::Invoke(detail);
		}
	};

	// Last LOG_THREADS
	template <int LOG_LOAD_VEC_SIZE, int LOG_LOADS_PER_TILE, int CACHE_MODIFIER, int WORK_STEALING>
	struct Iterate<MAX_LOG_THREADS, LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE, CACHE_MODIFIER, WORK_STEALING>
	{
		static void Invoke(Detail<T> &detail) {}
	};

	// Interface
	static void Invoke(Detail<T> &detail)
	{
		Iterate<MIN_LOG_THREADS, MIN_LOG_LOAD_VEC_SIZE, MIN_LOG_LOADS_PER_TILE, MIN_CACHE_MODIFIER, MIN_WORK_STEALING>::Invoke(detail);
	}
};




/******************************************************************************
 * MemcopyTuner
 ******************************************************************************/

class MemcopyTuner : public Architecture<__B40C_CUDA_ARCH__, MemcopyTuner>
{
	typedef Architecture<__B40C_CUDA_ARCH__, MemcopyTuner> 	BaseArchType;

	// Device properties
	const CudaProperties cuda_props;

public:

	// Constructor
	MemcopyTuner() {}

	// The arch version of the code for the current device that actually have
	// compiled kernels for
	int PtxVersion()
	{
		return cuda_props.kernel_ptx_version;
	}

	// Dispatch call-back with static CUDA_ARCH
	template <int CUDA_ARCH, typename Storage, typename Detail>
	hipError_t Enact(Storage &problem_storage, Detail &detail)
	{
		// Run the timing tests
		printf("\n");
		printf("sizeof(T), CTA_OCCUPANCY, LOG_THREADS, LOG_LOAD_VEC_SIZE, LOG_LOADS_PER_TILE, "
				"CACHE_MODIFIER, WORK_STEALING, LOG_SCHEDULE_GRANULARITY, "
				"elapsed time (ms), throughput (10^9 items/s), bandwidth (10^9 B/s)\n");
		SweepConfig<CUDA_ARCH, Storage>::Invoke(detail);

		return hipSuccess;
	}

	/**
	 * Creates an example memcopy problem and then dispatches the problem
	 * to the GPU for the given number of iterations, displaying runtime information.
	 *
	 * @param[in] 		num_elements
	 * 		Size in elements of the vector to copy
	 */
	template<typename T>
	void TestMemcopy(size_t num_elements)
	{
		printf("CodeGen: \t[device_sm_version: %d, kernel_ptx_version: %d]\n",
			cuda_props.device_sm_version, cuda_props.kernel_ptx_version);

		// Allocate the memcopy problem on the host and fill the keys with random bytes

		T *h_data 			= (T*) malloc(num_elements * sizeof(T));
		T *h_reference 		= (T*) malloc(num_elements * sizeof(T));

		if ((h_data == NULL) || (h_reference == NULL)){
			fprintf(stderr, "Host malloc of problem data failed\n");
			exit(1);
		}

		for (size_t i = 0; i < num_elements; ++i) {
//			RandomBits<T>(h_data[i], 0);
			h_data[i] = i;
			h_reference[i] = h_data[i];
		}

		printf("%d iterations, %d elements", g_iterations, num_elements);

		// Allocate device storage and enactor
		Detail<T> detail(num_elements);
		if (B40CPerror(hipMalloc((void**) &detail.d_src, sizeof(T) * num_elements),
			"TimedMemcopy hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);
		if (B40CPerror(hipMalloc((void**) &detail.d_dest, sizeof(T) * num_elements),
			"TimedMemcopy hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

		// Move a fresh copy of the problem into device storage
		if (B40CPerror(hipMemcpy(detail.d_src, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
			"TimedMemcopy hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);

		T dummy;
		BaseArchType::Enact(dummy, detail);

	    // Copy out data
	    if (B40CPerror(hipMemcpy(h_data, detail.d_dest, sizeof(T) * num_elements, hipMemcpyDeviceToHost),
			"TimedMemcopy hipMemcpy d_dest failed: ", __FILE__, __LINE__)) exit(1);

	    // Free allocated memory
	    if (detail.d_src) hipFree(detail.d_src);
	    if (detail.d_dest) hipFree(detail.d_dest);

		// Display copied data
		if (g_verbose) {
			printf("\n\nData:\n");
			for (int i = 0; i < num_elements; i++) {
				PrintValue<T>(h_data[i]);
				printf(", ");
			}
			printf("\n\n");
		}

	    // Verify solution
		CompareResults<T>(h_data, h_reference, num_elements, true);
		printf("\n");
		fflush(stdout);

		// Free our allocated host memory
		if (h_data) free(h_data);
	    if (h_reference) free(h_reference);
	}

};



/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

    int num_elements 					= 1024;

	// Check command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}

    args.GetCmdLineArgumenti("i", g_iterations);
    args.GetCmdLineArgumenti("n", num_elements);
    args.GetCmdLineArgumenti("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	MemcopyTuner tuner;

	// Execute test(s)
	tuner.TestMemcopy<unsigned char>(num_elements * 4);
//	tuner.TestMemcopy<unsigned short>(num_elements * 2);
//	tuner.TestMemcopy<unsigned int>(num_elements);
//	tuner.TestMemcopy<unsigned long long>(num_elements / 2);

	return 0;
}



