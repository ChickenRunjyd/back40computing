/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/

/******************************************************************************
 * BFS API: Expand-Contract, Single-Grid 
 ******************************************************************************/

#pragma once

#include "b40c_error_synchronize.cu"
#include "bfs_ec_sg_kernel.cu"

namespace b40c {


/**
 * A expand-then-contract, single-grid breadth-first-search (BFS-ECSG) enactor 
 * class.
 *  
 *   - Marks each node with its distance from the given "source" node.  (I.e., 
 *     nodes are marked with the iteration at which they were "discovered").
 *     
 *   - All iterations are performed by a single kernel-launch.  This is 
 *     made possible by software global-barriers across threadblocks.    
 * 
 * A BFS search iteratively expands outwards from the given source node.  At 
 * each iteration, the algorithm discovers unvisited nodes that are adjacent 
 * to the nodes discovered by the previous iteration.  The first iteration 
 * discovers the source node. 
 * 
 * This implementation uses a "expand-then-contract" approach for maintaining
 * a global queue of "frontier" nodes to inspect.  At each iteration, the 
 * frontier queue is comprised of "discovered nodes" from the previous 
 * iteration.  The algorithm expands these nodes into their edge-lists.  The
 * edges leading to previously-visited nodes are discarded.  Then the 
 * remaining (newly-discovered) nodes are enqueued into the frontier queue 
 * for the next iteration.
 * 
 */





}// namespace b40c

