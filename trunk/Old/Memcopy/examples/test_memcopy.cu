#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for *large-problem* memcopy.
 ******************************************************************************/

#include <stdio.h> 

// Memcopy includes
#include "memcopy_api_granularity.cuh"
#include "memcopy_api_enactor_tuned.cuh"

// Test utils
#include "b40c_util.h"

using namespace b40c;
using namespace memcopy;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool 	g_verbose 						= false;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;


/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\ntest_memcopy [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>] [--sweep]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the memcopy operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of bytes to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Timed memcopy.  Uses the GPU to copy the specified vector of elements for the given
 * number of iterations, displaying runtime information.
 *
 * @param[in] 		h_data
 * 		Vector of data to copy (also copied back out)
 */
template <typename T, ProblemSize PROBLEM_SIZE>
double TimedMemcopy(T *h_data, T *h_reference, size_t num_elements)
{
	printf("%d iterations, %d bytes\n\n", g_iterations, num_elements);
	
	// Allocate device storage  
	T *d_src, *d_dest;
	if (B40CPerror(hipMalloc((void**) &d_src, sizeof(T) * num_elements),
		"TimedMemcopy hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);
	if (B40CPerror(hipMalloc((void**) &d_dest, sizeof(T) * num_elements),
		"TimedMemcopy hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

	// Create enactor
	MemcopyEnactorTuned memcopy_enactor;

	// Move a fresh copy of the problem into device storage
	if (B40CPerror(hipMemcpy(d_src, h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
		"TimedMemcopy hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);

	// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
	memcopy_enactor.DEBUG = true;
	memcopy_enactor.template Enact<PROBLEM_SIZE>(
		d_dest, d_src, num_elements * sizeof(T), g_max_ctas);
	memcopy_enactor.DEBUG = false;

	// Perform the timed number of iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Start timing record
		hipEventRecord(start_event, 0);

		// Call the memcopy API routine
		memcopy_enactor.template Enact<PROBLEM_SIZE>(
			d_dest, d_src, num_elements * sizeof(T), g_max_ctas);

		// End timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;		
	}

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
    printf("\n%f GPU ms, %f x10^9 elts/sec, %f x10^9 B/sec\n",
		avg_runtime, throughput, 2 * throughput * sizeof(T));
	
    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);

    // Copy out data
    if (B40CPerror(hipMemcpy(h_data, d_dest, sizeof(T) * num_elements, hipMemcpyDeviceToHost),
		"TimedMemcopy hipMemcpy d_dest failed: ", __FILE__, __LINE__)) exit(1);
    
    // Free allocated memory
    if (d_src) hipFree(d_src);
    if (d_dest) hipFree(d_dest);

	// Flushes any stdio from the GPU
	hipDeviceSynchronize();

	// Display copied data
	if (g_verbose) {
		printf("\n\nData:\n");
		for (int i = 0; i < num_elements; i++) {
			PrintValue<T>(h_data[i]);
			printf(", ");
		}
		printf("\n\n");
	}

    // Verify solution
	CompareResults(h_data, h_reference, num_elements, true);
	printf("\n");
	fflush(stdout);

	return throughput;
}


/**
 * Creates an example memcopy problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<typename T>
void TestMemcopy(size_t num_elements)
{
    // Allocate the memcopy problem on the host and fill the keys with random bytes

	T *h_data 			= (T*) malloc(num_elements * sizeof(T));
	T *h_reference 		= (T*) malloc(num_elements * sizeof(T));

	if ((h_data == NULL) || (h_reference == NULL)){
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	for (size_t i = 0; i < num_elements; ++i) {
//		RandomBits<T>(h_data[i], 0);
		h_data[i] = i;
		h_reference[i] = h_data[i];
	}

	//
    // Run the timing test(s)
	//

	printf("\nUsing LARGE config: ");
	double large = TimedMemcopy<T, LARGE>(h_data, h_reference, num_elements);

	printf("\nUsing SMALL config: ");
	double small = TimedMemcopy<T, SMALL>(h_data, h_reference, num_elements);

	if (small > large) {
		printf("Small faster at %d bytes\n", num_elements);
	}

	// Free our allocated host memory 
	if (h_data) free(h_data);
    if (h_reference) free(h_reference);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

    //
	// Check command line arguments
    //

	size_t num_elements = 1024;

    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}

    bool sweep = args.CheckCmdLineFlag("sweep");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	// Execute test(s), optionally sweeping problem size downward
    size_t orig_num_elements = num_elements;
    do {

    	TestMemcopy<unsigned char>(num_elements);
		num_elements -= 4096;

    } while (sweep && (num_elements < orig_num_elements ));

	return 0;
}



