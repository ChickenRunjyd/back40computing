#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/

/******************************************************************************
 * API a the Single-grid BFS Imlementation
 ******************************************************************************/

#pragma once

#include <bfs_base.cu>
#include <bfs_common.cu>

#include <b40c/util/spine.cuh>
#include <b40c/bfs/problem_type.cuh>
#include <b40c/bfs/problem_config.cuh>

#include <b40c/bfs/expand_atomic/sweep_kernel.cuh>
#include <b40c/bfs/compact/upsweep_kernel.cuh>
#include <b40c/bfs/compact/downsweep_kernel.cuh>
#include <b40c/scan/spine_kernel.cuh>

namespace b40c {
namespace bfs {


/**
 * Level-grid breadth-first-search enactor.
 *  
 * Each iterations is performed by its own kernel-launch.  
 */
class LevelGridBfsEnactor : public BaseBfsEnactor
{
protected:
	
	/**
	 * Temporary device storage needed for reducing partials produced
	 * by separate CTAs
	 */
	util::Spine spine;

protected:

	/**
	 * Utility function: Returns the default maximum number of threadblocks 
	 * this enactor class can launch.
	 */
	int MaxGridSize(int cta_occupancy, int max_grid_size = 0)
	{
		if (max_grid_size <= 0) {
			// No override: Fully populate all SMs
			max_grid_size = this->cuda_props.device_props.multiProcessorCount * cta_occupancy;
		} 
		
		return max_grid_size;
	}
	
public: 	
	
	/**
	 * Constructor
	 */
	LevelGridBfsEnactor(bool DEBUG = false) : BaseBfsEnactor(DEBUG) {}


    /**
     * Obtain statistics about the last BFS search enacted 
     */
    void GetStatistics(
    	int &total_queued, 
    	int &passes, 
    	double &avg_barrier_wait)		// total time spent waiting in barriers in ms (threadblock average)
    {
    	total_queued = 0;
    	passes = 0;
    	avg_barrier_wait = 0;
    }
    
	/**
	 * Enacts a breadth-first-search on the specified graph problem.
	 *
	 * @return hipSuccess on success, error enumeration otherwise
	 */
    template <BfsStrategy STRATEGY, typename BfsCsrProblem>
	hipError_t EnactSearch(
		BfsCsrProblem 						&bfs_problem,
		typename BfsCsrProblem::VertexId 	src,
		int 								max_grid_size = 0)
	{
		// Compaction tuning configuration
		typedef ProblemConfig<
			typename BfsCsrProblem::ProblemType,
			200,
			util::io::ld::NONE,
			util::io::st::NONE,
			9,

			// Atomic expand
			6,
			8,
			7,
			0,
			0,
			5,
			true,

			// Compact upsweep
			8,
			7,
			0,
			0,

			// Compact spine
			5,
			2,
			0,
			5,

			// Compact downsweep
			8,
			7,
			1,
			1,
			5> ProblemConfig;

		typedef typename ProblemConfig::ExpandAtomicSweep 	ExpandAtomicSweep;
		typedef typename ProblemConfig::CompactUpsweep 		CompactUpsweep;
		typedef typename ProblemConfig::CompactSpine 		CompactSpine;
		typedef typename ProblemConfig::CompactDownsweep 	CompactDownsweep;

		typedef typename BfsCsrProblem::VertexId			VertexId;
		typedef typename BfsCsrProblem::SizeT				SizeT;


		hipError_t retval = hipSuccess;

		// Determine grid size
		int min_occupancy = B40C_MIN(CompactUpsweep::CTA_OCCUPANCY, B40C_MIN(CompactDownsweep::CTA_OCCUPANCY, ExpandAtomicSweep::CTA_OCCUPANCY));
		int grid_size = MaxGridSize(min_occupancy, max_grid_size);

		// Make sure our spine is big enough
		int spine_elements = grid_size;
		if (retval = spine.Setup<SizeT>(grid_size, spine_elements)) exit(1);


		printf("DEBUG: BFS min occupancy %d, level-grid size %d\n", min_occupancy, grid_size);

		VertexId iteration = 0;
		SizeT queue_length;

		while (true) {

			// BFS iteration
			expand_atomic::SweepKernel<ExpandAtomicSweep><<<grid_size, ExpandAtomicSweep::THREADS>>>(
				src,
				iteration,
				bfs_problem.d_queue[0],
				bfs_problem.d_queue[1],
				bfs_problem.d_column_indices,
				bfs_problem.d_row_offsets,
				bfs_problem.d_source_path,
				this->work_progress);

			iteration++;

			this->work_progress.GetQueueLength(iteration, queue_length);
			printf("Iteration %d BFS queued %lld elements\n",
				iteration - 1, (long long) queue_length);
			if (!queue_length) {
				break;
			}

			// Upsweep compact
			compact::UpsweepKernel<CompactUpsweep><<<grid_size, CompactUpsweep::THREADS>>>(
				iteration,
				bfs_problem.d_queue[1],
				bfs_problem.d_keep,
				(SizeT *) this->spine(),
				bfs_problem.d_collision_cache,
				this->work_progress);

			// Spine
			scan::SpineKernel<CompactSpine><<<1, CompactSpine::THREADS>>>(
				(SizeT*) spine(), (SizeT*) spine(), spine_elements);

			// Downsweep
			compact::DownsweepKernel<CompactDownsweep><<<grid_size, CompactDownsweep::THREADS>>>(
				iteration,
				bfs_problem.d_queue[1],
				bfs_problem.d_keep,
				bfs_problem.d_queue[0],
				(SizeT *) this->spine(),
				this->work_progress);
/*
			this->work_progress.GetQueueLength(iteration, queue_length);

			printf("Iteration %d compact queued %lld elements\n",
				iteration - 1, (long long) queue_length);

			if (!queue_length) {
				break;
			}
*/
		}





/*
		while (true) {

			// Contract-expand strategy
			BfsLevelGridKernel<VertexId, CollisionMask, CONTRACT_EXPAND><<<this->max_grid_size, CTA_THREADS>>>(
				src,
				bfs_problem.d_collision_cache,
				this->d_queue[queue_idx],
				this->d_queue[queue_idx ^ 1],
				bfs_problem.d_column_indices,
				bfs_problem.d_row_offsets,
				bfs_problem.d_source_path,
				this->d_queue_lengths,
				iteration);

			if (DEBUG && hipDeviceSynchronize()) {
				printf("BfsLevelGridKernel failed: %d %d", __FILE__, __LINE__);
				exit(1);
			}

			// Update out-queue length
			int outgoing_queue_length_idx = (iteration + 1) & 0x3;
			if (hipMemcpy(
				&num_elements,
				d_queue_lengths + outgoing_queue_length_idx,
				1 * sizeof(int),
				hipMemcpyDeviceToHost))
			{
				printf("hipMemcpy failed: %d %d", __FILE__, __LINE__);
				exit(1);
			}

			printf("Iteration %d output queued %d nodes\n", iteration, num_elements);

			if (num_elements == 0) {
				// No more work, all done.
				break;
			}

			queue_idx ^= 1;

			// Upsweep
			bfs::compact::UpsweepKernel<Upsweep><<<this->max_grid_size, Upsweep::THREADS>>>(
				this->d_queue[queue_idx],
				this->d_keep,
				(SizeT *) this->spine(),
				bfs_problem.d_collision_cache);

			// Spine
			scan::SpineKernel<Spine><<<1, Spine::THREADS>>>(
				(SizeT*) spine(), (SizeT*) spine(), spine_elements);

			// Downsweep
			bfs::compact::DownsweepKernel<Downsweep><<<this->max_grid_size, Downsweep::THREADS>>>(
				this->d_queue[queue_idx],
				this->d_keep,
				this->d_queue_lengths + outgoing_queue_length_idx,
				this->d_queue[queue_idx ^ 1],
				(SizeT *) this->spine());

			queue_idx ^= 1;

			iteration++;
		}
*/

		return retval;
	}
    
};




} // namespace bfs
} // namespace b40c

