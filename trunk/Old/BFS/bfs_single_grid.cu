/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/

/******************************************************************************
 * Level-grid BFS implementation
 ******************************************************************************/

#pragma once

#include <bfs_base.cu>

#include <b40c/util/spine.cuh>
#include <b40c/util/global_barrier.cuh>
#include <b40c/bfs/problem_type.cuh>
#include <b40c/bfs/single_grid/problem_config.cuh>
#include <b40c/bfs/single_grid/sweep_kernel.cuh>

#include <b40c/bfs/compact_expand/sweep_kernel_config.cuh>
#include <b40c/bfs/compact_expand/sweep_kernel.cuh>


namespace b40c {
namespace bfs {


/**
 * Single-grid breadth-first-search enactor.
 */
class SingleGridBfsEnactor : public BaseBfsEnactor
{

protected:

	/**
	 * Temporary device storage needed for reducing partials produced
	 * by separate CTAs
	 */
	util::Spine spine;

	/**
	 * Mechanism for implementing software global barriers from within
	 * a single grid invocation
	 */
	util::GlobalBarrierLifetime global_barrier;

public: 	
	
	/**
	 * Constructor
	 */
	SingleGridBfsEnactor(bool DEBUG = false) : BaseBfsEnactor(DEBUG) {}


    /**
     * Obtain statistics about the last BFS search enacted 
     */
	template <typename VertexId>
    void GetStatistics(
    	long long &total_queued,
    	VertexId &search_depth,
    	double &avg_barrier_wait)		// total time spent waiting in barriers in ms (threadblock average)
    {
    	total_queued = 0;
    	search_depth = 0;
    	avg_barrier_wait = 0;
    }
    
	/**
	 * Enacts a breadth-first-search on the specified graph problem.
	 *
	 * @return hipSuccess on success, error enumeration otherwise
	 */
    template <typename BfsCsrProblem>
	hipError_t EnactSearch(
		BfsCsrProblem 						&bfs_problem,
		typename BfsCsrProblem::VertexId 	src,
		int 								max_grid_size = 0)
	{
		hipError_t retval = hipSuccess;
		typedef typename BfsCsrProblem::SizeT SizeT;

		// Compaction tuning configuration
		typedef compact_expand::SweepKernelConfig<

				typename BfsCsrProblem::ProblemType,
				200,
				8,
				7,
				0,
				0,
				5,
				util::io::ld::cg,		// QUEUE_READ_MODIFIER,
				util::io::ld::NONE,		// COLUMN_READ_MODIFIER,
				util::io::ld::cg,		// ROW_OFFSET_ALIGNED_READ_MODIFIER,
				util::io::ld::NONE,		// ROW_OFFSET_UNALIGNED_READ_MODIFIER,
				util::io::st::cg,		// QUEUE_WRITE_MODIFIER,
				false,					// WORK_STEALING
				6> KernelConfig;

		int occupancy = KernelConfig::CTA_OCCUPANCY;
		int grid_size = MaxGridSize(occupancy, max_grid_size);

		printf("DEBUG: BFS occupancy %d, grid size %d\n",
			occupancy, grid_size);

		// Make sure spine and barriers are initialized
		int spine_elements = grid_size;
		if (retval = spine.Setup<SizeT>(grid_size, spine_elements)) exit(1);
		if (retval = global_barrier.Setup(grid_size)) (exit(1));

		fflush(stdout);

		compact_expand::SweepKernel<KernelConfig><<<grid_size, KernelConfig::THREADS>>>(
			src,

			bfs_problem.d_expand_queue,
			bfs_problem.d_expand_parent_queue,
			bfs_problem.d_compact_queue,
			bfs_problem.d_compact_parent_queue,

			bfs_problem.d_column_indices,
			bfs_problem.d_row_offsets,
			bfs_problem.d_source_path,
			bfs_problem.d_collision_cache,
			this->work_progress,
			this->global_barrier);

		if (retval = util::B40CPerror(hipDeviceSynchronize(),
			"SweepKernel failed", __FILE__, __LINE__)) exit(1);

		return retval;
	}
    
};




} // namespace bfs
} // namespace b40c

