/******************************************************************************
 * 
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 * 
 ******************************************************************************/

/******************************************************************************
 * Stub utilities for syncronizing after kernel launches (e.g., in debug 
 * versions) to catch errors and display device-generated stdout
 ******************************************************************************/

#pragma once

#include <stdio.h> 

namespace b40c {

/**
 * Block on the previous stream action (e.g., kernel launch), report error-status
 * and kernel-stdout if present 
 */
void synchronize(const char *message)
{
	hipError_t error = hipDeviceSynchronize();
	if(error) {
		fprintf(stderr, "%s caused %d (%s)\n", message, error, hipGetErrorString(error));
	}
} 

/**
 * Same as syncrhonize above, but conditional on definintion of __ERROR_SYNCHRONOUS
 */
void synchronize_if_enabled(const char *message)
{
#if defined(__ERROR_SYNCHRONOUS)
	synchronize(message);
#endif
} 



} // namespace b40c

