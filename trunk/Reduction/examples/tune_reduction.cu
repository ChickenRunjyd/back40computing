#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Tuning tool for establishing optimal reduction granularity configuration types
 ******************************************************************************/

#include <stdio.h> 

// Reduction includes
#include "reduction_api_granularity.cuh"
#include "reduction_api_enactor.cuh"

// Test utils
#include "b40c_util.h"
#include "b40c_numeric_traits.cuh"
#include "b40c_parameter_generation.cuh"

using namespace b40c;
using namespace traits;
using namespace reduction;

/******************************************************************************
 * Defines, constants, globals, and utility types
 ******************************************************************************/

bool g_verbose;
int g_max_ctas = 0;
int g_iterations = 0;


template <typename T>
struct Sum
{
	static __host__ __device__ __forceinline__ T Op(const T &a, const T &b)
	{
		return a + b;
	}

	static __host__ __device__ __forceinline__ T Identity()
	{
		return 0;
	}
};

template <typename T>
struct Max
{
	static __host__ __device__ __forceinline__ T Op(const T &a, const T &b)
	{
		return (a > b) ? a : b;
	}

	static __host__ __device__ __forceinline__ T Identity()
	{
		return 0;
	}
};



/******************************************************************************
 * Utility routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntune_reduction [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>]\n");
	printf("\n");
	printf("\t--v\tDisplays verbose configuration to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the reduction operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Timed reduction for applying a specific granularity configuration type
 */
template <typename TuneProblemDetail, typename Config>
void TimedReduction(TuneProblemDetail &detail)
{
	Config::Print();
	fflush(stdout);

	// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
	detail.enactor.DEBUG = g_verbose;

	if (detail.enactor.template Enact<Config>(detail.d_dest, detail.d_src, detail.num_elements, g_max_ctas)) {
		exit(1);
	}

	detail.enactor.DEBUG = false;

	// Perform the timed number of iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Start cuda timing record
		hipEventRecord(start_event, 0);

		// Call the reduction API routine
		if (detail.enactor.template Enact<Config>(detail.d_dest, detail.d_src, detail.num_elements, g_max_ctas)) {
			exit(1);
		}

		// End cuda timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;

		// Flushes any stdio from the GPU
		hipDeviceSynchronize();
	}

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput =  0.0;
	if (avg_runtime > 0.0) throughput = ((double) detail.num_elements) / avg_runtime / 1000.0 / 1000.0; 
    printf(", %f, %f, %f, ",
		avg_runtime, throughput, throughput * sizeof(typename TuneProblemDetail::T));
    fflush(stdout);

    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);

    // Copy out data
    if (B40CPerror(hipMemcpy(detail.h_data, detail.d_dest, sizeof(typename TuneProblemDetail::T), hipMemcpyDeviceToHost),
		"TimedReduction hipMemcpy d_dest failed: ", __FILE__, __LINE__)) exit(1);

    // Verify solution
	CompareResults<typename TuneProblemDetail::T>(detail.h_data, detail.h_reference, 1, true);
	printf("\n");
	fflush(stdout);

}


/******************************************************************************
 * Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

/**
 * Enumerated tuning params
 */
enum TuningParam {
	WORK_STEALING = 0,
	UNIFORM_SMEM_ALLOCATION,
	UNIFORM_GRID_SIZE,
	OVERSUBSCRIBED_GRID_SIZE,
	UPSWEEP_LOG_THREADS,
	UPSWEEP_LOG_LOAD_VEC_SIZE,
	UPSWEEP_LOG_LOADS_PER_TILE,

	PARAM_LIMIT,

	// Parameters below here are currently not part of the tuning sweep

	// These can be tuned, but we're currently not compelled to
	READ_MODIFIER,
	WRITE_MODIFIER,
	UPSWEEP_LOG_RAKING_THREADS,

	// Derive these from the others above
	UPSWEEP_CTA_OCCUPANCY,
	UPSWEEP_LOG_SCHEDULE_GRANULARITY,

	// General performance is insensitive to the spine kernel params
	// because it's only a single-CTA: we'll just use reasonable defaults
	SPINE_LOG_THREADS,
	SPINE_LOG_LOAD_VEC_SIZE,
	SPINE_LOG_LOADS_PER_TILE,
	SPINE_LOG_RAKING_THREADS
};


/**
 * Ranges for the tuning params
 */
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList, int PARAM> struct Ranges;

// READ_MODIFIER
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, READ_MODIFIER> {
	typedef typename ProblemDetail::T T;
	enum {
		MIN = NONE,
		MAX = ((CUDA_ARCH < 200) || (NumericTraits<T>::REPRESENTATION == NAN)) ? NONE : CS		// No type modifiers for pre-Fermi or non-builtin types
	};
};

// WRITE_MODIFIER
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, WRITE_MODIFIER> {
	typedef typename ProblemDetail::T T;
	enum {
		MIN = NONE,
		MAX = ((CUDA_ARCH < 200) || (NumericTraits<T>::REPRESENTATION == NAN)) ? NONE : CS		// No type modifiers for pre-Fermi or non-builtin types
	};
};

// WORK_STEALING
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, WORK_STEALING> {
	enum {
		MIN = 0,
		MAX = (CUDA_ARCH < 200) ? 0 : 1			// No workstealing pre-Fermi
	};
};

// UNIFORM_SMEM_ALLOCATION
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, UNIFORM_SMEM_ALLOCATION> {
	enum {
		MIN = 0,
		MAX = 1
	};
};

// UNIFORM_GRID_SIZE
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, UNIFORM_GRID_SIZE> {
	enum {
		MIN = 0,
		MAX = 1
	};
};

// OVERSUBSCRIBED_GRID_SIZE
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, OVERSUBSCRIBED_GRID_SIZE> {
	enum {
		MIN = 0,
		MAX = 1
	};
};

// UPSWEEP_LOG_THREADS
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, UPSWEEP_LOG_THREADS> {
	enum {
		MIN = B40C_LOG_WARP_THREADS(CUDA_ARCH),
		MAX = B40C_LOG_CTA_THREADS(CUDA_ARCH)
	};
};

// UPSWEEP_LOG_LOAD_VEC_SIZE
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, UPSWEEP_LOG_LOAD_VEC_SIZE> {
	enum {
		MIN = 0,
		MAX = 2
	};
};

// UPSWEEP_LOG_LOADS_PER_TILE
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, UPSWEEP_LOG_LOADS_PER_TILE> {
	enum {
		MIN = 0,
		MAX = 2
	};
};

// UPSWEEP_LOG_RAKING_THREADS
template <int CUDA_ARCH, typename ProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, ProblemDetail, ParamList, UPSWEEP_LOG_RAKING_THREADS> {
	enum {
		MIN = B40C_LOG_WARP_THREADS(CUDA_ARCH),
		MAX = ParamList::template Access<UPSWEEP_LOG_THREADS>::VALUE
	};
};



/******************************************************************************
 * Tuning Parameter Enumerations and Ranges
 ******************************************************************************/


/**
 * Encapsulation structure for
 * 		- Wrapping problem type and storage
 * 		- Providing call-back for parameter-list generation
 */
template <typename _T, typename _OpType>
struct TuneProblemDetail
{
	typedef _T T;
	typedef _OpType OpType;

	ReductionEnactor<> enactor;
	T *d_dest;
	T *d_src;
	T *h_data;
	T *h_reference;
	size_t num_elements;

	/**
	 * Constructor
	 */
	TuneProblemDetail(size_t num_elements) :
		d_dest(NULL), d_src(NULL), h_data(NULL), h_reference(NULL), num_elements(num_elements) {}

	/**
	 * Callback invoked by parameter-list generation
	 */
	template <int CUDA_ARCH, typename ParamList>
	void Invoke()
	{
		const int C_READ_MODIFIER =
//			ParamList::template Access<READ_MODIFIER>::VALUE;					// These can be tuned, but we're currently not compelled to
			NONE;
		const int C_WRITE_MODIFIER =
//			ParamList::template Access<WRITE_MODIFIER>::VALUE;					// These can be tuned, but we're currently not compelled to
			NONE;
		const int C_WORK_STEALING =
			ParamList::template Access<WORK_STEALING>::VALUE;
		const int C_UNIFORM_SMEM_ALLOCATION =
			ParamList::template Access<UNIFORM_SMEM_ALLOCATION>::VALUE;
		const int C_UNIFORM_GRID_SIZE =
			ParamList::template Access<UNIFORM_GRID_SIZE>::VALUE;
		const int C_OVERSUBSCRIBED_GRID_SIZE =
			ParamList::template Access<OVERSUBSCRIBED_GRID_SIZE>::VALUE;
		const int C_UPSWEEP_LOG_THREADS =
			ParamList::template Access<UPSWEEP_LOG_THREADS>::VALUE;
		const int C_UPSWEEP_LOG_LOAD_VEC_SIZE =
			ParamList::template Access<UPSWEEP_LOG_LOAD_VEC_SIZE>::VALUE;
		const int C_UPSWEEP_LOG_LOADS_PER_TILE =
			ParamList::template Access<UPSWEEP_LOG_LOADS_PER_TILE>::VALUE;
		const int C_UPSWEEP_LOG_RAKING_THREADS =
//			ParamList::template Access<UPSWEEP_LOG_RAKING_THREADS>::VALUE;		// These can be tuned, but we're currently not compelled to
			B40C_LOG_WARP_THREADS(CUDA_ARCH);

		const int C_UPSWEEP_CTA_OCCUPANCY = B40C_MIN(
			B40C_SM_CTAS(CUDA_ARCH),
			(B40C_SM_THREADS(CUDA_ARCH)) >> C_UPSWEEP_LOG_THREADS);
		const int C_UPSWEEP_LOG_SCHEDULE_GRANULARITY =
			C_UPSWEEP_LOG_LOADS_PER_TILE +
			C_UPSWEEP_LOG_LOAD_VEC_SIZE +
			C_UPSWEEP_LOG_THREADS;

		// General performance is insensitive to spine config it's only a single-CTA:
		// simply use reasonable defaults
		const int C_SPINE_LOG_THREADS = 8;
		const int C_SPINE_LOG_LOAD_VEC_SIZE = 0;
		const int C_SPINE_LOG_LOADS_PER_TILE = 1;
		const int C_SPINE_LOG_RAKING_THREADS = B40C_LOG_WARP_THREADS(CUDA_ARCH);
		
		// Establish the problem type
		typedef ReductionProblem<
			typename TuneProblemDetail::T,
			size_t,
			TuneProblemDetail::OpType::Op,
			TuneProblemDetail::OpType::Identity> Problem;

		// Establish the granularity configuration type
		typedef ReductionConfig <Problem,
			(CacheModifier) C_READ_MODIFIER,
			(CacheModifier) C_WRITE_MODIFIER,
			C_WORK_STEALING,
			C_UNIFORM_SMEM_ALLOCATION,
			C_UNIFORM_GRID_SIZE,
			C_OVERSUBSCRIBED_GRID_SIZE,
			C_UPSWEEP_CTA_OCCUPANCY,
			C_UPSWEEP_LOG_THREADS,
			C_UPSWEEP_LOG_LOAD_VEC_SIZE,
			C_UPSWEEP_LOG_LOADS_PER_TILE,
			C_UPSWEEP_LOG_RAKING_THREADS,
			C_UPSWEEP_LOG_SCHEDULE_GRANULARITY,
			C_SPINE_LOG_THREADS, 
			C_SPINE_LOG_LOAD_VEC_SIZE, 
			C_SPINE_LOG_LOADS_PER_TILE, 
			C_SPINE_LOG_RAKING_THREADS> Config;		

		// Invoke this config
		TimedReduction<TuneProblemDetail, Config>(*this);
	}
};



/**
 * Reduction Tuner
 */
class ReductionTuner : public Architecture<__B40C_CUDA_ARCH__, ReductionTuner>
{
	typedef Architecture<__B40C_CUDA_ARCH__, ReductionTuner> 	BaseArchType;

	// Device properties
	const CudaProperties cuda_props;

public:

	// Constructor
	ReductionTuner() {}

	// Return the current device's sm version
	int PtxVersion()
	{
		return cuda_props.device_sm_version;
	}

	// Dispatch call-back with static CUDA_ARCH
	template <int CUDA_ARCH, typename Storage, typename TuneProblemDetail>
	hipError_t Enact(Storage &problem_storage, TuneProblemDetail &detail)
	{
		// Run the timing tests
		ParamListSweep<
			CUDA_ARCH,
			TuneProblemDetail,
			0,
			PARAM_LIMIT,
			Ranges>::template Invoke<void>(detail);
		return hipSuccess;
	}

	/**
	 * Creates an example reduction problem and then dispatches the problem
	 * to the GPU for the given number of iterations, displaying runtime information.
	 */
	template<typename T, typename OpType>
	void TestReduction(size_t num_elements)
	{
		// Allocate storage and enactor
		TuneProblemDetail<T, OpType> detail(num_elements);

		if (B40CPerror(hipMalloc((void**) &detail.d_src, sizeof(T) * num_elements),
			"TimedReduction hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);

		if (B40CPerror(hipMalloc((void**) &detail.d_dest, sizeof(T)),
			"TimedReduction hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

		if ((detail.h_data = (T*) malloc(num_elements * sizeof(T))) == NULL) {
			fprintf(stderr, "Host malloc of problem data failed\n");
			exit(1);
		}
		if ((detail.h_reference = (T*) malloc(sizeof(T))) == NULL) {
			fprintf(stderr, "Host malloc of problem data failed\n");
			exit(1);
		}

		detail.h_reference[0] = OpType::Identity();
		for (size_t i = 0; i < num_elements; ++i) {
			// RandomBits<T>(detail.h_data[i], 0);
			detail.h_data[i] = i;
			detail.h_reference[0] = OpType::Op(detail.h_reference[0], detail.h_data[i]);
		}

		// Move a fresh copy of the problem into device storage
		if (B40CPerror(hipMemcpy(detail.d_src, detail.h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
			"TimedReduction hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);

		// Have the base class call back with a constant dispatch-arch for our current device
		T dummy;
		BaseArchType::Enact(dummy, detail);

	    // Free allocated memory
	    if (detail.d_src) hipFree(detail.d_src);
	    if (detail.d_dest) hipFree(detail.d_dest);

		// Free our allocated host memory
		if (detail.h_data) free(detail.h_data);
	    if (detail.h_reference) free(detail.h_reference);
	}

};



/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

    size_t num_elements 								= 1024;

	// Check command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}

    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	CudaProperties cuda_props;

	printf("Test Reduction: %d iterations, %d elements", g_iterations, num_elements);
	printf("\nCodeGen: \t[device_sm_version: %d, kernel_ptx_version: %d]\n\n",
		cuda_props.device_sm_version, cuda_props.kernel_ptx_version);

	printf("READ_MODIFIER, WRITE_MODIFIER, WORK_STEALING, UNIFORM_SMEM_ALLOCATION, UNIFORM_GRID_SIZE, OVERSUBSCRIBED_GRID_SIZE, "
		"UPSWEEP_CTA_OCCUPANCY, UPSWEEP_LOG_THREADS, UPSWEEP_LOG_LOAD_VEC_SIZE, UPSWEEP_LOG_LOADS_PER_TILE, UPSWEEP_LOG_RAKING_THREADS, UPSWEEP_LOG_SCHEDULE_GRANULARITY, "
		"SPINE_LOG_THREADS, SPINE_LOG_LOAD_VEC_SIZE, SPINE_LOG_LOADS_PER_TILE, SPINE_LOG_RAKING_THREADS, "
		"elapsed time (ms), throughput (10^9 items/s), bandwidth (10^9 B/s), Correctness\n");

	ReductionTuner tuner;

//	typedef unsigned char T;
//	typedef unsigned short T;
	typedef unsigned int T;
//	typedef unsigned long long T;

	// Execute test(s)
	tuner.TestReduction<T, Sum<T> >(num_elements * sizeof(num_elements) / 4);

	return 0;
}



