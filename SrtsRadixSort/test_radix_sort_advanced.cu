#include "hip/hip_runtime.h"
/**
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * 
 * 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 */


//------------------------------------------------------------------------------
// Advanced test driver program for SRTS Radix Sorting
//
// WARNING: This program assumes knowlege of the temporary storage management
// needed for performing SRTS radix sort -- do not use it as a reference for 
// embedding SRTS sorting within your application.  See the Simple test driver 
// program instead.
//------------------------------------------------------------------------------

#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <math.h> 
#include <float.h>

#include <inc/cutil.h>


//------------------------------------------------------------------------------
// Sorting includes
//------------------------------------------------------------------------------

#include <srts_radix_sort.cu>			// Sorting includes
#include <test_radix_sort_utils.cu>		// Utilities and correctness-checking


//------------------------------------------------------------------------------
// Defines, constants, globals 
//------------------------------------------------------------------------------

bool g_verbose;
bool g_verbose2;
bool g_verify;
int  g_entropy_reduction = 0;


//------------------------------------------------------------------------------
// Empty Kernels
//------------------------------------------------------------------------------

/**
 * Dummy kernel to demarcate iterations of the same problem size in the profiler logs 
 */
__global__ void DummyKernel()
{
}



//------------------------------------------------------------------------------
// Routines
//------------------------------------------------------------------------------

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\nsrts_radix_sort [--device=<device index>] [--v[2]] [--noverify]\n");
	printf("[--i=<num-iterations>] [--entropy-reduction=<level>]\n");
	printf("[--key-bytes=<1|2|4|8>] [--value-bytes=<0|4|8|16>]\n");
	printf("[--n=<num-elements> | --n-input=<num-elements listfile>]\n");
	printf("[--max-blocks=<max-thread-blocks> | --max-blocks-input=<max-thread-blocks listfile>]\n");
	printf("\n");
	printf("\t--v\tDisplays kernel launch config info.\n");
	printf("\n");
	printf("\t--v2\tSame as --v, but displays the sorted keys to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the sorting operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
	printf("\t--n-input\tA file of problem sizes, one per line.\n");
	printf("\n");
	printf("\t--max-blocks\tThe maximum number of threadblocks to launch.\n");
	printf("\t\t\tDefault = -1 (i.e., the API will select an appropriate value)\n");
	printf("\n");
	printf("\t--max-blocks-input\tA file of maximum threadblocks, one per line.\n");
	printf("\n");
	printf("\t--key-bytes\tThe number of key bytes to use.  Default=4\n");
	printf("\n");
	printf("\t--value-bytes\tThe number of value satellite bytes to pair with\n");
	printf("\t\t\tthe key.  Default=0 (I.e., keys-only)\n");
	printf("\n");
	printf("[\t--entropy-reduction=<level>\tSpecifies the number of bitwise-AND'ing\n");
	printf("\t\t\titerations for random key data.  Default = 0, Identical keys = -1\n");
	printf("\n");
	printf("\t--noverify\tSpecifies that results should not be copied back and checked for correctness\n");
	printf("\n");
}


/**
 * Reads a newline-separated list of numbers from an input file.
 * Allocates memory for the returned list.
 */
void ReadList(
	int* &list, 
	unsigned int &len, 
	char* filename, 
	unsigned int default_val) 
{
	if (filename == NULL) {
		len = 1;
		list = (int*) malloc(len * sizeof(int));
		list[0] = default_val;
		return;
	}

	unsigned int data;
	FILE* fin = fopen(filename, "r");
	if (fin == NULL) {
		fprintf(stderr, "Could not open file.  Exiting.\n");
		exit(1);
	}
	len = 0;

	while(fscanf(fin, "%d\n", &data) > 0) {
		len++;
	}

	list = (int*) malloc(len * sizeof(int));
	rewind(fin);
	len = 0;

	while(fscanf(fin, "%d\n", &data) > 0) {
	
		list[len] = data;
		len++;
	}

	fclose(fin);
}


/**
 * Returns whether or not the problem will fit on the device.
 */
template <typename K, typename V>
bool CanFit(hipDeviceProp_t &device_props, bool keys_only, unsigned long long problem_size) {
	
	long long bytes = problem_size * sizeof(K) * 2;
	if (!keys_only) bytes += problem_size * sizeof(V) * 2;
	return (bytes < ((double) device_props.totalGlobalMem) * 0.90); 	// allow up to 90% capacity 
}


/**
 * Uses the GPU to sort the specified vector of elements for the given 
 * number of iterations, displaying runtime information.
 */
template <typename K, typename V>
void TimedSort(
	unsigned int num_elements, 
	unsigned int max_grid_size,
	K *h_keys,
	GlobalStorage<K, V>	&device_storage,
	unsigned int iterations,
	bool keys_only) 
{
	CUT_CHECK_ERROR("Kernel execution failed (errors before launch)");

	// Create timing records
	hipEvent_t start_event, stop_event;
	CUDA_SAFE_CALL( hipEventCreate(&start_event) );
	CUDA_SAFE_CALL( hipEventCreate(&stop_event) );

	// Perform the timed number of sorting iterations
	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < iterations; i++) {

		SRTS_DEBUG = (g_verbose && (i == 0));

		// Move a fresh copy of the problem into device storage
		CUDA_SAFE_CALL( hipMemcpy(device_storage.keys, h_keys, num_elements * sizeof(K), hipMemcpyHostToDevice) );

		// Start cuda timing record
		CUDA_SAFE_CALL( hipEventRecord(start_event, 0) );

		// Call the sorting API routine
		LaunchKeyValueSort<K, V>(num_elements, device_storage, max_grid_size);

		// End cuda timing record
		CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
		CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
		CUDA_SAFE_CALL( hipEventElapsedTime(&duration, start_event, stop_event));
		elapsed += (double) duration;
		
		if (i == 0) {
			printf("%d-byte keys, %d-byte values, %d iterations, %d elements", 
				sizeof(K), 
				(keys_only) ? 0 : sizeof(V),
				iterations, 
				num_elements);
			fflush(stdout);
		}
	}

	// Display timing information
	double avg_runtime = elapsed / iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0; 
    printf(", %f GPU ms, %f x10^9 elts/sec\n", 
		avg_runtime,
		throughput);

    // Clean up events
	CUDA_SAFE_CALL( hipEventDestroy(start_event) );
	CUDA_SAFE_CALL( hipEventDestroy(stop_event) );
	
	// Copy out sorted keys and check
    if (g_verify || g_verbose) {

    	CUDA_SAFE_CALL( hipMemcpy(h_keys, device_storage.keys, num_elements * sizeof(K), hipMemcpyDeviceToHost) );

		// Display sorted key data
		if (g_verbose2) {
			printf("\n\nKeys:\n");
			for (int i = 0; i < num_elements; i++) {	
				PrintValue<K>(h_keys[i]);
				printf(", ");
			}
			printf("\n\n");
		}	
		
	    // Verify solution
		if (g_verify) {
			VerifySort<K>(h_keys, num_elements, true);
			printf("\n");
			fflush(stdout);
		}
    }
	
}


/**
 * Creates an example sorting problem whose keys is a vector of the specified 
 * number of K elements, values of V elements, and then dispatches the problem 
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<typename K, typename V>
void TestSort(
	bool keys_only,
	unsigned int iterations,
	int* problem_sizes,
	unsigned int num_problem_sizes,
	int* max_grid_sizes,
	unsigned int num_max_grid_sizes) 
{
	unsigned int radix_bits = 4;
	
    GlobalStorage<K, V> device_storage = {NULL, NULL, NULL, NULL, NULL};
	K* h_keys;

	// Get device properties
	int current_device;
	hipDeviceProp_t device_props;
	hipGetDevice(&current_device);
	hipGetDeviceProperties(&device_props, current_device);
	unsigned int sm_version = device_props.major * 100 + device_props.minor * 10;
	unsigned int cycle_elements = SRTS_CYCLE_ELEMENTS(sm_version, K, V);
	
	// find maximum problem size in the list of problem sizes
	unsigned int max_problem_size = 0;
	for (int i = 0; i < num_problem_sizes; i++) {
		if ((problem_sizes[i] > max_problem_size) && CanFit<K, V>(device_props, keys_only, problem_sizes[i])) {
			max_problem_size = problem_sizes[i];
		}
	}
	
	// Allocate device memory
	h_keys = (K*) malloc(max_problem_size * sizeof(K));
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.keys, max_problem_size * sizeof(K)) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.temp_keys, max_problem_size * sizeof(K)));
	if (!keys_only) {
		CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.data, max_problem_size * sizeof(V)));
		CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.temp_data, max_problem_size * sizeof(V)));
	}

	// Find largest maximum grid size in list of maximum grid sizes
	int max_grid_size = -1;
	for (int i = 0; i < num_max_grid_sizes; i++) {
		if (max_grid_sizes[i] > max_grid_size) {
			max_grid_size = max_grid_sizes[i];
		}
	}

	// Allocate device vector for holding the spine
	unsigned int max_spine_len = GridSize(max_problem_size, max_grid_size, cycle_elements, device_props, sm_version);   
	max_spine_len *= (1 << radix_bits);																						// multiply by number of histogram digits  
	max_spine_len = ((max_spine_len + SRTS_SPINE_CYCLE_ELEMENTS - 1) / SRTS_SPINE_CYCLE_ELEMENTS) * SRTS_SPINE_CYCLE_ELEMENTS;	// round up to nearest cycle size
	CUDA_SAFE_CALL( hipMalloc( (void**) &device_storage.temp_spine, max_spine_len * sizeof(unsigned int)) );

	// Run combinations of specified problem-sizes & max-grid-sizes
	for (int i = 0; i < num_problem_sizes; i++) {

		if (!CanFit<K, V>(device_props, keys_only, problem_sizes[i])) {
			printf("Problem size %d too large\n", problem_sizes[i]);
			continue;
		}

		// Randomly initialize the keyset on the host
		for (unsigned int j = 0; j < problem_sizes[i]; j++) {
			RandomBits<K>(h_keys[j], g_entropy_reduction);
		}
		
		for (int j = 0; j < num_max_grid_sizes; j++) {

			// Run a dummy kernel to demarcate the start of this set of iterations in the counter logs
			DummyKernel<<<1,1,0>>>();

			// Run the timing test 
			TimedSort<K, V>(problem_sizes[i], max_grid_sizes[j], h_keys, device_storage, iterations, keys_only);
		}
	}
    
    // cleanup memory
	free(h_keys);
	CUDA_SAFE_CALL(hipFree(device_storage.keys));
	CUDA_SAFE_CALL(hipFree(device_storage.temp_keys));
	CUDA_SAFE_CALL(hipFree(device_storage.temp_spine));
	if (!keys_only) {
		CUDA_SAFE_CALL(hipFree(device_storage.data));
		CUDA_SAFE_CALL(hipFree(device_storage.temp_data));
	}	
	
}


template<typename K>
void TestSort(
	int value_bytes,
	unsigned int iterations,
	int* problem_sizes,
	unsigned int num_problem_sizes,
	int* max_grid_sizes,
	unsigned int num_max_grid_sizes)
{
	switch (value_bytes) {
	case 0:		// keys only
		TestSort<K, unsigned int>((value_bytes == 0), iterations, problem_sizes, num_problem_sizes, max_grid_sizes, num_max_grid_sizes);
		break;
	case 4:		// 32-bit values
		TestSort<K, unsigned int>((value_bytes == 0), iterations, problem_sizes, num_problem_sizes, max_grid_sizes, num_max_grid_sizes);
		break;
	case 8:		// 64-bit values
		TestSort<K, unsigned long long>((value_bytes == 0), iterations, problem_sizes, num_problem_sizes, max_grid_sizes, num_max_grid_sizes);
		break;
	case 16:	// 128-bit values
		TestSort<K, uint4>((value_bytes == 0), iterations, problem_sizes, num_problem_sizes, max_grid_sizes, num_max_grid_sizes);
		break;
	default: 
		fprintf(stderr, "Invalid payload size.  Exiting.\n");
	}
}



//------------------------------------------------------------------------------
// Main
//------------------------------------------------------------------------------

int main( int argc, char** argv) {

	CUT_DEVICE_INIT(argc, argv);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

	unsigned int num_elements 				= 512;
    int max_grid_size 						= -1;	// let API determine best grid size
	unsigned int iterations  				= 1;
	char *problem_sizes_filename 			= NULL;
	char *max_grid_sizes_filename 			= NULL;
	int key_bytes							= 4;
	int value_bytes							= 0;
	int* problem_sizes 						= NULL;
	int* max_grid_sizes 					= NULL;
	unsigned int num_problem_sizes;
	unsigned int num_max_grid_sizes;

    //
	// Check command line arguments
    //

    if (cutCheckCmdLineFlag( argc, (const char**) argv, "help")) {
		Usage();
		return 0;
	}

    cutGetCmdLineArgumenti( argc, (const char**) argv, "i", (int*)&iterations);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "key-bytes", (int*)&key_bytes);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "value-bytes", (int*)&value_bytes);
	cutGetCmdLineArgumentstr( argc, (const char**) argv, "n-input", &problem_sizes_filename);
	cutGetCmdLineArgumenti( argc, (const char**) argv, "max-blocks", (int*)&max_grid_size);
	cutGetCmdLineArgumentstr( argc, (const char**) argv, "max-blocks-input", &max_grid_sizes_filename);
	cutGetCmdLineArgumenti( argc, (const char**) argv, "entropy-reduction", (int*)&g_entropy_reduction);
	if (g_verbose2 = cutCheckCmdLineFlag( argc, (const char**) argv, "v2")) {
		g_verbose = true;
	} else {
		g_verbose = cutCheckCmdLineFlag( argc, (const char**) argv, "v");
	}
	g_verify = !cutCheckCmdLineFlag( argc, (const char**) argv, "noverify");
	
	// Attempt to read list of problem sizes to run
	ReadList(
		problem_sizes, 
		num_problem_sizes, 
		problem_sizes_filename, 
		num_elements); 

	// Attempt to read list of max-grid-sizes to run
	ReadList(
		max_grid_sizes, 
		num_max_grid_sizes, 
		max_grid_sizes_filename, 
		max_grid_size); 
	
	// Execute test(s)

	switch (key_bytes) {
	case 1:		// 8-bit keys
		TestSort<unsigned char>(value_bytes, iterations, problem_sizes, num_problem_sizes, max_grid_sizes, num_max_grid_sizes);
		break;
	case 2:		// 16-bit keys
		TestSort<unsigned short>(value_bytes, iterations, problem_sizes, num_problem_sizes, max_grid_sizes, num_max_grid_sizes);
		break;
	case 4:		// 32-bit keys
		TestSort<unsigned int>(value_bytes, iterations, problem_sizes, num_problem_sizes, max_grid_sizes, num_max_grid_sizes);
		break;
	case 8:		// 64-bit keys
		TestSort<unsigned long long>(value_bytes, iterations, problem_sizes, num_problem_sizes, max_grid_sizes, num_max_grid_sizes);
		break;
	default: 
		fprintf(stderr, "Invalid key size.  Exiting.\n");
	}
}



