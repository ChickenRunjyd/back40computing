#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Tuning tool for establishing optimal scan granularity configuration types
 ******************************************************************************/

#include <stdio.h> 

// Scan includes
#include <b40c/arch_dispatch.cuh>
#include <b40c/scan/granularity.cuh>
#include <b40c/scan_enactor.cuh>
#include <b40c/util/cuda_properties.cuh>
#include <b40c/util/numeric_traits.cuh>
#include <b40c/util/parameter_generation.cuh>

// Test utils
#include "b40c_util.h"

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals, and utility types
 ******************************************************************************/

#ifndef TUNE_ARCH
	#define TUNE_ARCH (200)
#endif

bool g_verbose;
int g_max_ctas = 0;
int g_iterations = 0;


template <typename T>
struct Sum
{
	static __host__ __device__ __forceinline__ T Op(const T &a, const T &b)
	{
		return a + b;
	}

	static __host__ __device__ __forceinline__ T Identity()
	{
		return 0;
	}
};

template <typename T>
struct Max
{
	static __host__ __device__ __forceinline__ T Op(const T &a, const T &b)
	{
		return (a > b) ? a : b;
	}

	static __host__ __device__ __forceinline__ T Identity()
	{
		return 0;
	}
};



/******************************************************************************
 * Utility routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntune_scan [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>]\n");
	printf("\n");
	printf("\t--v\tDisplays verbose configuration to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the scan operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Timed scan for applying a specific granularity configuration type
 */
template <typename TuneProblemDetail, typename Config>
void TimedScan(TuneProblemDetail &detail)
{
	typedef typename TuneProblemDetail::T T;

	printf("%d, ", sizeof(T));
	Config::Print();
	fflush(stdout);

	// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
	detail.enactor.DEBUG = g_verbose;
	if (detail.enactor.template Enact<Config>(detail.d_dest, detail.d_src, detail.num_elements, g_max_ctas)) {
		exit(1);
	}
	detail.enactor.DEBUG = false;

	// Perform the timed number of iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Start cuda timing record
		hipEventRecord(start_event, 0);

		// Call the scan API routine
		if (detail.enactor.template Enact<Config>(detail.d_dest, detail.d_src, detail.num_elements, g_max_ctas)) {
			exit(1);
		}

		// End cuda timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;

		// Flushes any stdio from the GPU
		hipDeviceSynchronize();
	}

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput =  0.0;
	if (avg_runtime > 0.0) throughput = ((double) detail.num_elements) / avg_runtime / 1000.0 / 1000.0; 
    printf(", %f, %f, %f, ",
		avg_runtime, throughput, throughput * sizeof(typename TuneProblemDetail::T));
    fflush(stdout);

    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);

    // Copy out data
    if (util::B40CPerror(hipMemcpy(detail.h_data, detail.d_dest, sizeof(T) * detail.num_elements, hipMemcpyDeviceToHost),
		"TimedScan hipMemcpy d_dest failed: ", __FILE__, __LINE__)) exit(1);

    // Verify solution
	CompareResults<typename TuneProblemDetail::T>(detail.h_data, detail.h_reference, 1, true);
	printf("\n");
	fflush(stdout);

}


/******************************************************************************
 * Tuning Parameter Enumerations and Ranges
 ******************************************************************************/

/**
 * Enumerated tuning params
 */
enum TuningParam {
	UNIFORM_SMEM_ALLOCATION,
	UNIFORM_GRID_SIZE,
	OVERSUBSCRIBED_GRID_SIZE,

	UPSWEEP_LOG_THREADS,
	UPSWEEP_LOG_LOAD_VEC_SIZE,
	UPSWEEP_LOG_LOADS_PER_TILE,

	DOWNSWEEP_LOG_THREADS,
	DOWNSWEEP_LOG_LOAD_VEC_SIZE,
	DOWNSWEEP_LOG_LOADS_PER_TILE,

	PARAM_LIMIT,

	// Parameters below here are currently not part of the tuning sweep

	// These can be tuned, but we're currently not compelled to
	READ_MODIFIER,
	WRITE_MODIFIER,
	UPSWEEP_LOG_RAKING_THREADS,
	DOWNSWEEP_LOG_RAKING_THREADS,

	// Derive these from the others above
	UPSWEEP_CTA_OCCUPANCY,
	DOWNSWEEP_CTA_OCCUPANCY,
	LOG_SCHEDULE_GRANULARITY,

	// General performance is insensitive to the spine kernel params
	// because it's only a single-CTA: we'll just use reasonable defaults
	SPINE_LOG_THREADS,
	SPINE_LOG_LOAD_VEC_SIZE,
	SPINE_LOG_LOADS_PER_TILE,
	SPINE_LOG_RAKING_THREADS
};


/**
 * Ranges for the tuning params
 */
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList, int PARAM> struct Ranges;

// READ_MODIFIER
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, READ_MODIFIER> {
	typedef typename TuneProblemDetail::T T;
	enum {
		MIN = util::ld::NONE,
		MAX = ((CUDA_ARCH < 200) || (util::NumericTraits<T>::REPRESENTATION == util::NAN)) ? util::ld::NONE : util::ld::CS		// No type modifiers for pre-Fermi or non-builtin types
	};
};

// WRITE_MODIFIER
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, WRITE_MODIFIER> {
	typedef typename TuneProblemDetail::T T;
	enum {
		MIN = util::st::NONE,
		MAX = ((CUDA_ARCH < 200) || (util::NumericTraits<T>::REPRESENTATION == util::NAN)) ? util::st::NONE : util::st::CS		// No type modifiers for pre-Fermi or non-builtin types
	};
};

// UNIFORM_SMEM_ALLOCATION
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, UNIFORM_SMEM_ALLOCATION> {
	enum {
		MIN = 0,
		MAX = 1
	};
};

// UNIFORM_GRID_SIZE
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, UNIFORM_GRID_SIZE> {
	enum {
		MIN = 0,
		MAX = 1
	};
};

// OVERSUBSCRIBED_GRID_SIZE
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, OVERSUBSCRIBED_GRID_SIZE> {
	enum {
		MIN = 0,
		MAX = 1
	};
};

// UPSWEEP_LOG_THREADS
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, UPSWEEP_LOG_THREADS> {
	enum {
		MIN = B40C_LOG_WARP_THREADS(CUDA_ARCH),
		MAX = B40C_LOG_CTA_THREADS(CUDA_ARCH)
	};
};

// UPSWEEP_LOG_LOAD_VEC_SIZE
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, UPSWEEP_LOG_LOAD_VEC_SIZE> {
	enum {
		MIN = 0,
		MAX = 2
	};
};

// UPSWEEP_LOG_LOADS_PER_TILE
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, UPSWEEP_LOG_LOADS_PER_TILE> {
	enum {
		MIN = 0,
		MAX = 2
	};
};

// UPSWEEP_LOG_RAKING_THREADS
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, UPSWEEP_LOG_RAKING_THREADS> {
	enum {
		MIN = B40C_LOG_WARP_THREADS(CUDA_ARCH),
		MAX = ParamList::template Access<UPSWEEP_LOG_THREADS>::VALUE
	};
};

// DOWNSWEEP_LOG_THREADS
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, DOWNSWEEP_LOG_THREADS> {
	enum {
		MIN = B40C_LOG_WARP_THREADS(CUDA_ARCH),
		MAX = B40C_LOG_CTA_THREADS(CUDA_ARCH)
	};
};

// DOWNSWEEP_LOG_LOAD_VEC_SIZE
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, DOWNSWEEP_LOG_LOAD_VEC_SIZE> {
	enum {
		MIN = 0,
		MAX = 2
	};
};

// DOWNSWEEP_LOG_LOADS_PER_TILE
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, DOWNSWEEP_LOG_LOADS_PER_TILE> {
	enum {
		MIN = 0,
		MAX = 2
	};
};

// DOWNSWEEP_LOG_RAKING_THREADS
template <int CUDA_ARCH, typename TuneProblemDetail, typename ParamList>
struct Ranges<CUDA_ARCH, TuneProblemDetail, ParamList, DOWNSWEEP_LOG_RAKING_THREADS> {
	enum {
		MIN = B40C_LOG_WARP_THREADS(CUDA_ARCH),
		MAX = ParamList::template Access<DOWNSWEEP_LOG_THREADS>::VALUE
	};
};


/******************************************************************************
 * Tuning Parameter Enumerations and Ranges
 ******************************************************************************/


/**
 * Encapsulation structure for
 * 		- Wrapping problem type and storage
 * 		- Providing call-back for parameter-list generation
 */
template <typename _T, typename _OpType>
struct TuneProblemDetail
{
	typedef _T T;
	typedef _OpType OpType;

	ScanEnactor<> enactor;
	T *d_dest;
	T *d_src;
	T *h_data;
	T *h_reference;
	size_t num_elements;

	/**
	 * Constructor
	 */
	TuneProblemDetail(size_t num_elements) :
		d_dest(NULL), d_src(NULL), h_data(NULL), h_reference(NULL), num_elements(num_elements) {}

	/**
	 * Callback invoked by parameter-list generation
	 */
	template <int CUDA_ARCH, typename ParamList>
	void Invoke()
	{
		const int C_READ_MODIFIER =
//			ParamList::template Access<READ_MODIFIER>::VALUE;					// These can be tuned, but we're currently not compelled to
			util::ld::NONE;
		const int C_WRITE_MODIFIER =
//			ParamList::template Access<WRITE_MODIFIER>::VALUE;					// These can be tuned, but we're currently not compelled to
			util::ld::NONE;
		const int C_UNIFORM_SMEM_ALLOCATION =
			ParamList::template Access<UNIFORM_SMEM_ALLOCATION>::VALUE;
		const int C_UNIFORM_GRID_SIZE =
			ParamList::template Access<UNIFORM_GRID_SIZE>::VALUE;
		const int C_OVERSUBSCRIBED_GRID_SIZE =
			ParamList::template Access<OVERSUBSCRIBED_GRID_SIZE>::VALUE;

		const int C_UPSWEEP_LOG_THREADS =
			ParamList::template Access<UPSWEEP_LOG_THREADS>::VALUE;
		const int C_UPSWEEP_LOG_LOAD_VEC_SIZE =
			ParamList::template Access<UPSWEEP_LOG_LOAD_VEC_SIZE>::VALUE;
		const int C_UPSWEEP_LOG_LOADS_PER_TILE =
			ParamList::template Access<UPSWEEP_LOG_LOADS_PER_TILE>::VALUE;
		const int C_UPSWEEP_LOG_RAKING_THREADS =
//			ParamList::template Access<UPSWEEP_LOG_RAKING_THREADS>::VALUE;		// These can be tuned, but we're currently not compelled to
			B40C_LOG_WARP_THREADS(CUDA_ARCH);
		const int C_UPSWEEP_CTA_OCCUPANCY = B40C_MIN(
			B40C_SM_CTAS(CUDA_ARCH),
			(B40C_SM_THREADS(CUDA_ARCH)) >> C_UPSWEEP_LOG_THREADS);

		const int C_DOWNSWEEP_LOG_THREADS =
			ParamList::template Access<DOWNSWEEP_LOG_THREADS>::VALUE;
		const int C_DOWNSWEEP_LOG_LOAD_VEC_SIZE =
			ParamList::template Access<DOWNSWEEP_LOG_LOAD_VEC_SIZE>::VALUE;
		const int C_DOWNSWEEP_LOG_LOADS_PER_TILE =
			ParamList::template Access<DOWNSWEEP_LOG_LOADS_PER_TILE>::VALUE;
		const int C_DOWNSWEEP_LOG_RAKING_THREADS =
//			ParamList::template Access<DOWNSWEEP_LOG_RAKING_THREADS>::VALUE;		// These can be tuned, but we're currently not compelled to
			B40C_LOG_WARP_THREADS(CUDA_ARCH);
		const int C_DOWNSWEEP_CTA_OCCUPANCY = B40C_MIN(
			B40C_SM_CTAS(CUDA_ARCH),
			(B40C_SM_THREADS(CUDA_ARCH)) >> C_DOWNSWEEP_LOG_THREADS);


		const int C_UPSWEEP_LOG_SCHEDULE_GRANULARITY =
			C_UPSWEEP_LOG_LOADS_PER_TILE +
			C_UPSWEEP_LOG_LOAD_VEC_SIZE +
			C_UPSWEEP_LOG_THREADS;

		const int C_DOWNSWEEP_LOG_SCHEDULE_GRANULARITY =
			C_DOWNSWEEP_LOG_LOADS_PER_TILE +
			C_DOWNSWEEP_LOG_LOAD_VEC_SIZE +
			C_DOWNSWEEP_LOG_THREADS;

		// TODO: figure out if we should use min here instead
		const int C_LOG_SCHEDULE_GRANULARITY = B40C_MAX(
			C_UPSWEEP_LOG_SCHEDULE_GRANULARITY,
			C_DOWNSWEEP_LOG_SCHEDULE_GRANULARITY);

		// General performance is insensitive to spine config it's only a single-CTA:
		// simply use reasonable defaults
		const int C_SPINE_LOG_THREADS = 8;
		const int C_SPINE_LOG_LOAD_VEC_SIZE = 0;
		const int C_SPINE_LOG_LOADS_PER_TILE = 1;
		const int C_SPINE_LOG_RAKING_THREADS = B40C_LOG_WARP_THREADS(CUDA_ARCH);
		
		// Establish the problem type
		typedef scan::ScanProblemType<
			typename TuneProblemDetail::T,
			size_t,
			TuneProblemDetail::OpType::Op,
			TuneProblemDetail::OpType::Identity> ScanProblemType;

		// Establish the granularity configuration type
		typedef scan::ScanConfig <ScanProblemType,
			(util::ld::CacheModifier) C_READ_MODIFIER,
			(util::st::CacheModifier) C_WRITE_MODIFIER,
			C_UNIFORM_SMEM_ALLOCATION,
			C_UNIFORM_GRID_SIZE,
			C_OVERSUBSCRIBED_GRID_SIZE,
			C_LOG_SCHEDULE_GRANULARITY,

			C_UPSWEEP_CTA_OCCUPANCY,
			C_UPSWEEP_LOG_THREADS,
			C_UPSWEEP_LOG_LOAD_VEC_SIZE,
			C_UPSWEEP_LOG_LOADS_PER_TILE,
			C_UPSWEEP_LOG_RAKING_THREADS,

			C_SPINE_LOG_THREADS, 
			C_SPINE_LOG_LOAD_VEC_SIZE, 
			C_SPINE_LOG_LOADS_PER_TILE, 
			C_SPINE_LOG_RAKING_THREADS,

			C_DOWNSWEEP_CTA_OCCUPANCY,
			C_DOWNSWEEP_LOG_THREADS,
			C_DOWNSWEEP_LOG_LOAD_VEC_SIZE,
			C_DOWNSWEEP_LOG_LOADS_PER_TILE,
			C_DOWNSWEEP_LOG_RAKING_THREADS> ScanConfig;

		// Invoke this config
		TimedScan<TuneProblemDetail, ScanConfig>(*this);
	}
};


/**
 * Creates an example scan problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<typename T, typename OpType>
void TestScan(size_t num_elements)
{
	// Allocate storage and enactor
	typedef TuneProblemDetail<T, OpType> Detail;
	Detail detail(num_elements);

	if (util::B40CPerror(hipMalloc((void**) &detail.d_src, sizeof(T) * num_elements),
		"TimedScan hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);

	if (util::B40CPerror(hipMalloc((void**) &detail.d_dest, sizeof(T)),
		"TimedScan hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

	if ((detail.h_data = (T*) malloc(num_elements * sizeof(T))) == NULL) {
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}
	if ((detail.h_reference = (T*) malloc(sizeof(T))) == NULL) {
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	detail.h_reference[0] = OpType::Identity();
	for (size_t i = 0; i < num_elements; ++i) {
		// RandomBits<T>(detail.h_data[i], 0);
		detail.h_data[i] = i;
		detail.h_reference[0] = OpType::Op(detail.h_reference[0], detail.h_data[i]);
	}

	// Move a fresh copy of the problem into device storage
	if (util::B40CPerror(hipMemcpy(detail.d_src, detail.h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
		"TimedScan hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);

	// Run the timing tests
	util::ParamListSweep<
		TUNE_ARCH,
		Detail,
		0,
		PARAM_LIMIT,
		Ranges>::template Invoke<void>(detail);

	// Free allocated memory
	if (detail.d_src) hipFree(detail.d_src);
	if (detail.d_dest) hipFree(detail.d_dest);

	// Free our allocated host memory
	if (detail.h_data) free(detail.h_data);
	if (detail.h_reference) free(detail.h_reference);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{

	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

    size_t num_elements 								= 1024;

	// Check command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}

    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	util::CudaProperties cuda_props;

	printf("Test Scan: %d iterations, %d elements", g_iterations, num_elements);
	printf("\nCodeGen: \t[device_sm_version: %d, kernel_ptx_version: %d]\n\n",
		cuda_props.device_sm_version, cuda_props.kernel_ptx_version);

	printf("sizeof(T), READ_MODIFIER, WRITE_MODIFIER, UNIFORM_SMEM_ALLOCATION, UNIFORM_GRID_SIZE, OVERSUBSCRIBED_GRID_SIZE, LOG_SCHEDULE_GRANULARITY, "
		"UPSWEEP_CTA_OCCUPANCY, UPSWEEP_LOG_THREADS, UPSWEEP_LOG_LOAD_VEC_SIZE, UPSWEEP_LOG_LOADS_PER_TILE, UPSWEEP_LOG_RAKING_THREADS, "
		"SPINE_LOG_THREADS, SPINE_LOG_LOAD_VEC_SIZE, SPINE_LOG_LOADS_PER_TILE, SPINE_LOG_RAKING_THREADS, "
		"DOWNSWEEP_CTA_OCCUPANCY, DOWNSWEEP_LOG_THREADS, DOWNSWEEP_LOG_LOAD_VEC_SIZE, DOWNSWEEP_LOG_LOADS_PER_TILE, DOWNSWEEP_LOG_RAKING_THREADS, "
		"elapsed time (ms), throughput (10^9 items/s), bandwidth (10^9 B/s), Correctness\n");

	// Execute test(s)
/*
	{
		typedef unsigned char T;
		TestScan<T, Sum<T> >(num_elements * 4);
	}
	{
		typedef unsigned short T;
		TestScan<T, Sum<T> >(num_elements * 2);
	}
*/
	{
		typedef unsigned int T;
		TestScan<T, Sum<T> >(num_elements);
	}
/*
	{
		typedef unsigned long long T;
		TestScan<T, Sum<T> >(num_elements / 2);
	}
*/

	return 0;
}



