/******************************************************************************
 * 
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 * 
 ******************************************************************************/

/******************************************************************************
 * CUDA Properties
 ******************************************************************************/

#pragma once

#include <hip/hip_runtime.h>

namespace b40c {

// CUDA architecture currently being compiled for
#ifndef __CUDA_ARCH__
	#define __B40C_CUDA_ARCH__ 0						// Host
#else
	#define __B40C_CUDA_ARCH__ __CUDA_ARCH__
#endif

// Thread per warp. (The CUDA Toolkit gives us warp-size, but not the log of it, which is also useful)
#define B40C_LOG_WARP_THREADS(arch)		(5)			// 32 threads in a warp 
#define B40C_WARP_THREADS(arch)			(1 << B40C_LOG_WARP_THREADS(arch))

// Memory banks per SM
#define B40C_SM20_LOG_MEM_BANKS()		(5)			// 32 banks on SM2.0+
#define B40C_SM10_LOG_MEM_BANKS()		(4)			// 16 banks on SM1.0-SM1.3
#define B40C_LOG_MEM_BANKS(arch)		((arch >= 200) ? B40C_SM20_LOG_MEM_BANKS() : 	\
															 B40C_SM10_LOG_MEM_BANKS())		

// Physical shared memory per SM (bytes)
#define B40C_SM20_SMEM_BYTES()			(49152)		// 48KB on SM2.0+
#define B40C_SM10_SMEM_BYTES()			(16384)		// 32KB on SM1.0-SM1.3
#define B40C_SMEM_BYTES(arch)			((arch >= 200) ? B40C_SM20_SMEM_BYTES() : 	\
														 B40C_SM10_SMEM_BYTES())		

// Physical threads per SM (bytes)
#define B40C_SM20_SM_THREADS()			(1536)		// 1536 threads on SM2.0+
#define B40C_SM12_SM_THREADS()			(1024)		// 1024 threads on SM1.2-SM1.3
#define B40C_SM10_SM_THREADS()			(768)		// 768 threads on SM1.0-SM1.1
#define B40C_SM_THREADS(arch)			((arch >= 200) ? B40C_SM20_SMEM_BYTES() : 	\
										 (arch >= 200) ? B40C_SM12_SMEM_BYTES() : 	\
														 B40C_SM10_SMEM_BYTES())		

// Register modifier for pointer-types (for inlining PTX assembly)
#if defined(_WIN64) || defined(__LP64__)
	#define _B40C_LP64_ true			
	// 64-bit register modifier for inlined asm
	#define _B40C_ASM_PTR_ "l"
#else
	#define _B40C_LP64_ false
	// 32-bit register modifier for inlined asm
	#define _B40C_ASM_PTR_ "r"
#endif


/**
 * Empty Kernel
 */
template <typename T>
__global__ void FlushKernel(void) { }


/**
 * Class encapsulating device properties for dynamic host-side inspection
 */
class CudaProperties 
{
public:
	
	// Information about our target device
	hipDeviceProp_t 		device_props;
	int 				device_sm_version;
	
	// Information about our kernel assembly
	int 				kernel_ptx_version;
	
public:
	
	CudaProperties() 
	{
		// Get current device properties 
		int current_device;
		hipGetDevice(&current_device);
		hipGetDeviceProperties(&device_props, current_device);
		device_sm_version = device_props.major * 100 + device_props.minor * 10;
	
		// Get SM version of compiled kernel assemblies
		hipFuncAttributes flush_kernel_attrs;
		hipFuncGetAttributes(&flush_kernel_attrs, reinterpret_cast<const void*>(FlushKernel<void>));
		kernel_ptx_version = flush_kernel_attrs.ptxVersion * 10;
	}
};


} // namespace b40c

