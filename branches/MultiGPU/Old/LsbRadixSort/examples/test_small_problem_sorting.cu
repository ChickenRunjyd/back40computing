/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * 
 * 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for *small-problem* radix sorting (with an 
 * optionally-reduced number of valid key-bits).
 *
 * Useful for demonstrating how to integrate LsbSingleGrid radix sorting into 
 * your application 
 ******************************************************************************/

#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <math.h> 
#include <float.h>

// Sorting includes
#include <radixsort_single_grid.cu>
#include <radixsort_early_exit.cu>		

#include <test_utils.cu>				// Utilities and correctness-checking
#include <cutil.h>						// Utilities for commandline parsing
#include <b40c_util.h>					// Misc. utils (random-number gen, I/O, etc.)

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

//#define __B40C_ERROR_CHECKING__		 

bool g_verbose;


/******************************************************************************
 * Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\ntest_small_problem_sorting [--device=<device index>] [--v] [--i=<num-iterations>] [--n=<num-elements>] [--keys-only]\n"); 
	printf("\n");
	printf("\t--v\tDisplays sorted results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the sorting operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}



/**
 * Uses the small-problem-sorter (single-grid enactor) sort the specified 17-bit sorting
 * problem whose keys is a vector of the specified number of unsigned int elements, 
 * values of unsigned int elements.
 *
 * @param[in] 		num_elements 
 * 		Size in elements of the vector to sort
 * @param[in] 		h_keys 
 * 		Vector of keys to sort 
 * @param[in] 		iterations  
 * 		Number of times to invoke the GPU sorting primitive
 */
template <typename K, typename V, int LOWER_KEY_BITS> 
void SmallProblemTimedSort(
	unsigned int num_elements, 
	K *h_keys,
	K *h_reference_keys,
	unsigned int iterations)
{
	printf("Single-kernel, small-problem key-value sort, %d iterations, %d elements", iterations, num_elements);
	
	// Allocate device storage   
	MultiCtaRadixSortStorage<K, V> device_storage(num_elements);	
	hipMalloc((void**) &device_storage.d_keys[0], sizeof(K) * num_elements);
    dbg_perror_exit("SmallProblemTimedSort:: hipMalloc device_storage.d_keys[0] failed: ", __FILE__, __LINE__);
    hipMalloc((void**) &device_storage.d_values[0], sizeof(V) * num_elements);
    dbg_perror_exit("SmallProblemTimedSort:: hipMalloc device_storage.d_values[0] failed: ", __FILE__, __LINE__);

	// Create sorting enactor
	SingleGridRadixSortingEnactor<K, V> sorting_enactor;

	// Perform a single sorting iteration to allocate memory, prime code caches, etc.
	hipMemcpy(
		device_storage.d_keys[0], 
		h_keys, 
		sizeof(K) * num_elements, 
		hipMemcpyHostToDevice);											// copy keys
	sorting_enactor.template EnactSort<LOWER_KEY_BITS>(device_storage);		// sort

	// Perform the timed number of sorting iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < iterations; i++) {

		sorting_enactor.RADIXSORT_DEBUG = (i == 0);

		// Move a fresh copy of the problem into device storage
		hipMemcpy(
			device_storage.d_keys[0], 
			h_keys, 
			sizeof(K) * num_elements, 
			hipMemcpyHostToDevice);										// copy keys

		// Start cuda timing record
		hipEventRecord(start_event, 0);

		// Call the sorting API routine
		sorting_enactor.template EnactSort<LOWER_KEY_BITS>(device_storage);	// sort

		// End cuda timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;		
	}

	// Display timing information
	double avg_runtime = elapsed / iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0; 
    printf(", %f GPU ms, %f x10^9 elts/sec\n", 
		avg_runtime,
		throughput);
	
    // Copy out keys 
    hipMemcpy(
    	h_keys, 
    	device_storage.d_keys[device_storage.selector], 
    	sizeof(K) * num_elements, 
    	hipMemcpyDeviceToHost);
    
    // Free allocated memory
    if (device_storage.d_keys[0]) hipFree(device_storage.d_keys[0]);
    if (device_storage.d_keys[1]) hipFree(device_storage.d_keys[1]);
    if (device_storage.d_values[0]) hipFree(device_storage.d_values[0]);
    if (device_storage.d_values[1]) hipFree(device_storage.d_values[1]);

    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
	
	// Display sorted key data
	if (g_verbose) {
		printf("\n\nKeys:\n");
		for (int i = 0; i < num_elements; i++) {	
			PrintValue<K>(h_keys[i]);
			printf(", ");
		}
		printf("\n\n");
	}	
	
    // Verify solution
	CompareResults<K>(h_keys, h_reference_keys, num_elements, true);
	printf("\n");
	fflush(stdout);
}



/**
 * Uses the large-problem-sorter (early-exit enactor) sort the specified sorting
 * problem whose keys is a vector of the specified number of K elements, 
 * values of V elements.
 *
 * @param[in] 		num_elements 
 * 		Size in elements of the vector to sort
 * @param[in] 		h_keys 
 * 		Vector of keys to sort 
 * @param[in] 		iterations  
 * 		Number of times to invoke the GPU sorting primitive
 */
template <typename K, typename V, int LOWER_KEY_BITS> 
void LargeProblemTimedSort(
	unsigned int num_elements, 
	K *h_keys,
	K *h_reference_keys,
	unsigned int iterations)
{
	printf("Early-exit key-value sort, %d iterations, %d elements", iterations, num_elements);
	
	// Allocate device storage   
	MultiCtaRadixSortStorage<K, V> device_storage(num_elements);	
	hipMalloc((void**) &device_storage.d_keys[0], sizeof(K) * num_elements);
    dbg_perror_exit("LargeProblemTimedSort:: hipMalloc device_storage.d_keys[0] failed: ", __FILE__, __LINE__);
	hipMalloc((void**) &device_storage.d_values[0], sizeof(V) * num_elements);
    dbg_perror_exit("LargeProblemTimedSort:: hipMalloc device_storage.d_values[0] failed: ", __FILE__, __LINE__);

	// Create sorting enactor
	EarlyExitRadixSortingEnactor<K, V> sorting_enactor;

	// Perform a single sorting iteration to allocate memory, prime code caches, etc.
	hipMemcpy(
		device_storage.d_keys[0], 
		h_keys, 
		sizeof(K) * num_elements, 
		hipMemcpyHostToDevice);		// copy keys
	sorting_enactor.EnactSort(device_storage);

	// Perform the timed number of sorting iterations

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	double elapsed = 0;
	float duration = 0;
	for (int i = 0; i < iterations; i++) {

		sorting_enactor.RADIXSORT_DEBUG = (i == 0);

		// Move a fresh copy of the problem into device storage
		hipMemcpy(
			device_storage.d_keys[0], 
			h_keys, 
			sizeof(K) * num_elements, 
			hipMemcpyHostToDevice);		// copy keys

		// Start cuda timing record
		hipEventRecord(start_event, 0);

		// Call the sorting API routine
		sorting_enactor.EnactSort(device_storage);

		// End cuda timing record
		hipEventRecord(stop_event, 0);
		hipEventSynchronize(stop_event);
		hipEventElapsedTime(&duration, start_event, stop_event);
		elapsed += (double) duration;		
	}

	// Display timing information
	double avg_runtime = elapsed / iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0; 
    printf(", %f GPU ms, %f x10^9 elts/sec\n", 
		avg_runtime,
		throughput);
	
    // Copy out keys 
    hipMemcpy(
    	h_keys, 
    	device_storage.d_keys[device_storage.selector], 
    	sizeof(K) * num_elements, 
    	hipMemcpyDeviceToHost);
    
    // Free allocated memory
    if (device_storage.d_keys[0]) hipFree(device_storage.d_keys[0]);
    if (device_storage.d_keys[1]) hipFree(device_storage.d_keys[1]);
    if (device_storage.d_values[0]) hipFree(device_storage.d_values[0]);
    if (device_storage.d_values[1]) hipFree(device_storage.d_values[1]);

    // Clean up events
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);
	
	// Display sorted key data
	if (g_verbose) {
		printf("\n\nKeys:\n");
		for (int i = 0; i < num_elements; i++) {	
			PrintValue<K>(h_keys[i]);
			printf(", ");
		}
		printf("\n\n");
	}	
	
    // Verify solution
	CompareResults<K>(h_keys, h_reference_keys, num_elements, true);
	printf("\n");
	fflush(stdout);
}


/**
 * Creates an example sorting problem whose keys is a vector of the specified 
 * number of elements having the specfied number of valid bits, and then 
 * dispatches the problem to the GPU for the given number of iterations, 
 * displaying runtime information.
 *
 * @param[in] 		iterations  
 * 		Number of times to invoke the GPU sorting primitive
 * @param[in] 		num_elements 
 * 		Size in elements of the vector to sort
 * @param[in]		use_small_problem_enactor
 */
template <typename K, typename V, int LOWER_KEY_BITS> 
void TestSort(
	unsigned int iterations,
	int num_elements) 
{
    // Allocate the sorting problem on the host and fill the keys with random bytes

	K *h_keys = NULL;
	K *h_reference_keys = NULL;
	h_keys = (K*) malloc(num_elements * sizeof(K));
	h_reference_keys = (K*) malloc(num_elements * sizeof(K));

	// Use random bits
	for (unsigned int i = 0; i < num_elements; ++i) {
		RandomBits<K>(h_keys[i], 0, LOWER_KEY_BITS);
		h_reference_keys[i] = h_keys[i];
	}

	// Sort the reference keys
	std::sort(h_reference_keys, h_reference_keys + num_elements);	

	//
    // Run the timing tests
	//
	
	// Single-grid enactor (explicit passes)
	SmallProblemTimedSort<K, V, LOWER_KEY_BITS>(
		num_elements, h_keys, h_reference_keys, iterations);

	// Early-exit enactor (dynamic pass detection)
	LargeProblemTimedSort<K, V, LOWER_KEY_BITS>(
		num_elements, h_keys, h_reference_keys, iterations);

	// Free our allocated host memory 
	if (h_keys != NULL) free(h_keys);
	if (h_reference_keys != NULL) free(h_reference_keys);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main( int argc, char** argv) {

	CUT_DEVICE_INIT(argc, argv);

	//srand(time(NULL));	
	srand(0);				// presently deterministic

    unsigned int num_elements 					= 1024;
    unsigned int iterations  					= 1;

	// Check command line arguments
    if (cutCheckCmdLineFlag( argc, (const char**) argv, "help")) {
		Usage();
		return 0;
	}
    cutGetCmdLineArgumenti( argc, (const char**) argv, "i", (int*)&iterations);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);
	g_verbose = cutCheckCmdLineFlag( argc, (const char**) argv, "v");

	// Run sorting examples
	
	TestSort<unsigned int, unsigned int, 17>(			// only sort lower 17 bits 
		iterations, num_elements);

/*	
	TestSort<float, float, sizeof(float) * 8>(
		iterations, num_elements);	
	TestSort<long long, long long, sizeof(long long) * 8>(
		iterations, num_elements);
	TestSort<char, char, sizeof(char) * 8>(
		iterations, num_elements);
	TestSort<double, double, sizeof(double) * 8>(
		iterations, num_elements);
*/		
	
	hipDeviceSynchronize();
}



