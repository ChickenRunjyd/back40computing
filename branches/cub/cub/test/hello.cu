

#include <stdio.h>


/**
 * Main
 */
int main(int argc, const char**argv)
{
	typedef int T;

	hipError_t error;

	// Set device
	int current_device = 0;
	if (argc > 1) {
		current_device = atoi(argv[1]);
	}
	if (error = hipSetDevice(current_device)) {
		printf("hipSetDevice failed (%d:%s)\n",  error, hipGetErrorString(error));
		exit(1);
	}

	hipDeviceProp_t device_props;
	if (error = hipGetDeviceProperties(&device_props, current_device)) {
		printf("hipGetDeviceProperties failed (%d:%s)\n",  error, hipGetErrorString(error));
		exit(1);
	}
	printf("Device(%s), UVA(%d)\n", device_props.name, device_props.unifiedAddressing);

	// Allocate device data
	T *d_data;
	if (error = hipMalloc((void**) &d_data, sizeof(T))) {
		printf("hipMalloc failed (%d:%s)\n",  error, hipGetErrorString(error));
		exit(1);
	}

	hipPointerAttribute_t pointer_attrs;
	if (error = hipPointerGetAttributes(&pointer_attrs, d_data)) {
		printf("cudaPointerGetAttributes1 failed (%d:%s)\n",  error, hipGetErrorString(error));
		exit(1);
	}

	printf("Type(%s), device(%d)\n",
		(pointer_attrs.memoryType == hipMemoryTypeDevice) ? "Device" : "Host",
		pointer_attrs.device);

	int h_data[5];

	if (error = hipPointerGetAttributes(&pointer_attrs, h_data)) {
		printf("cudaPointerGetAttributes2 failed (%d:%s)\n",  error, hipGetErrorString(error));
		exit(1);
	}

	printf("Type(%s), device(%d)\n",
		(pointer_attrs.memoryType == hipMemoryTypeDevice) ? "Device" : "Host",
		pointer_attrs.device);

	return 0;
}
